#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.
// Copyright (c) 2004-2008 AGEIA Technologies, Inc. All rights reserved.
// Copyright (c) 2001-2004 NovodeX AG. All rights reserved.  

#include "CuSolverKernel.h"
#include "CuClothData.h"
#include "CuPhaseConfig.h" 

#include <new> // placement new

using namespace physx;

#if __CUDA_ARCH__ < 200
#define printf(x)
#define assert(x) /* not supported */
#else
extern "C"
{
	extern _CRTIMP __host__ __device__ int __cdecl printf(const char*, ...);
}
#endif 

// global symbol holding all cloth instances
static __constant__ cloth::CuKernelData gKernelData;

// shared memory copy (instead of relying on constant cache)
__shared__ cloth::CuClothData gClothData; 
__shared__ cloth::CuFrameData gFrameData; 
__shared__ cloth::CuIterationData gIterData;

static const uint32_t gCuClothDataSize = sizeof(cloth::CuClothData) / sizeof(float);
static const uint32_t gCuFrameDataSize = sizeof(cloth::CuFrameData) / sizeof(float);
static const uint32_t gCuIterationDataSize = sizeof(cloth::CuIterationData) / sizeof(float);
static const uint32_t gCuPhaseConfigSize = sizeof(cloth::CuPhaseConfig) / sizeof(float);

/*
Memory block for all temporary data in shared memory (in 'allocation' order).
The numbers indicate the allocation slot if used a stack allocator.
0) simulate*()::configs (numPhases*sizeof(CuPhaseConfig))
1) simulate*()::particles ({0,1,2}*4*numParticles floats)
2) CuCollision::mCapsuleIndices, mCapsuleMasks, mConvexMasks (numCapsules*4+numConvexes ints)
3) CuCollision::mPrevData (4*numSpheres+10*numCones floats)
4) CuCollision::collideConvexes() (4*numPlanes floats)
4) CuCollision::collideTriangles() (19*numTriangles floats)
4) CuCollision::mCurData::Spheres (4*numSpheres floats)
5) computeParticleBounds()::dst (192 floats written, 208 float read)
5) computeSphereBounds()::dst (192 floats written, 208 floats read)
5) CuCollision::mCurData::Cones (10*numCones floats)
6) CuCollision::mShapeGrid (2*6*sGridSize=96 floats)
4) CuSelfCollision::buildAcceleration()::buffer (34*16=544 ints)
*/ 
extern __shared__ float gSharedMemory[];
extern __shared__ uint32_t gSharedUnsigned[];

#if __CUDA_ARCH__ < 200

__device__ float* sharedBase(const float&) { return gSharedMemory; }
__device__ uint32_t* sharedBase(const uint32_t&) { return gSharedUnsigned; }

// pointer forced to point to shared memory (only works for sizeof(T) <= 4)
template <typename T>
class SharedPointer
{
	__device__ explicit SharedPointer(ptrdiff_t offset) : mOffset(offset) {}

public:
	typedef SharedPointer<T> Type;

	__device__ SharedPointer() {}
	__device__ SharedPointer(const SharedPointer& other) : mOffset(other.mOffset) {}
	__device__ SharedPointer(T* ptr) : mOffset(ptr-sharedBase(T())) {}

	template <typename S> // assuming pointee types of convertible pointers have same size
	__device__ SharedPointer(const SharedPointer<S>& other) : mOffset(other.mOffset) {}

	__device__ bool operator!=(const SharedPointer& other) const { return mOffset != other.mOffset; }
	__device__ bool operator<(const SharedPointer& other) const { return mOffset < other.mOffset; }

	__device__ SharedPointer operator+(ptrdiff_t i) const { return SharedPointer(mOffset + i); }
	__device__ SharedPointer& operator+=(ptrdiff_t i) { mOffset += i; return *this; }
	__device__ SharedPointer operator-(ptrdiff_t i) const { return SharedPointer(mOffset - i); }

	__device__ SharedPointer& operator++() { ++mOffset; return *this; }
	__device__ SharedPointer& operator--() { --mOffset; return *this; }

	__device__ SharedPointer operator++(int) { return SharedPointer(mOffset++); }

	__device__ T* operator->() const { return sharedBase(T()) + mOffset; }
	__device__ T& operator*() const { return sharedBase(T())[mOffset]; }
	__device__ T& operator[](int32_t i) const { return sharedBase(T())[mOffset+i]; }

	ptrdiff_t mOffset;

	PX_COMPILE_TIME_ASSERT(sizeof(T) == 4);
};

#else
template <typename T>
struct SharedPointer
{
	typedef T* Type;
};
#endif

// pointer with stride of 4
template <typename T>
struct GlobalPointer
{
	typedef GlobalPointer<T> Type;

	__device__ GlobalPointer() {}
	__device__ GlobalPointer(const GlobalPointer& other) : mPtr(other.mPtr) {}
	__device__ GlobalPointer(T* ptr) : mPtr(ptr) {}

	template <typename S> 
	__device__ GlobalPointer(const GlobalPointer<S>& other) 
		: mPtr(other.mPtr) 
	{}

	__device__ bool operator<(const GlobalPointer& other) const { return mPtr < other.mPtr; }

	__device__ GlobalPointer operator+(int32_t i) const { return GlobalPointer(mPtr + 4*i); }
	__device__ GlobalPointer& operator+=(int32_t i) { mPtr += 4*i; return *this; }

	__device__ T& operator*() const { return *mPtr; }
	__device__ T& operator[](uint32_t i) const { return mPtr[4*i]; }

	T* mPtr;
};

template <template <typename> class PointerT>
struct ParticleData
{
	typedef typename PointerT<float>::Type PointerType;
	typedef typename PointerT<const float>::Type ConstPointerType;

	__device__ ParticleData() {}	// Empty constructor required for -G0 flag

	__device__ float& operator()(uint32_t index, uint32_t element)
	{
		return mPointers[element][index];
	}
	__device__ const float& operator()(uint32_t index, uint32_t element) const
	{
		return mPointers[element][index];
	}

	__device__ const PointerType& operator[](uint32_t element) 
	{
		return mPointers[element]; 
	}
	__device__ ConstPointerType operator[](uint32_t element) const 
	{
		return mPointers[element]; 
	}

	PointerType mPointers[4];
};

/***************** Profiling **********************/
struct ProfileDisabledZone 
{ 
	__device__ ProfileDisabledZone(cloth::CuProfileZoneIds::Enum) {} 
};


#if defined(__CUDA_ARCH__) && defined(PX_PROFILE) // profile zones enabled for profile build

#include "GPUProfile.h"

struct ProfileZone
{
	__device__ ProfileZone(cloth::CuProfileZoneIds::Enum id)
		: mEvent(0)
	{
		if (!gKernelData.mProfileBuffer || threadIdx.x & 0x1f)
			return;

		// +1: first entry reserved for counter
		uint32_t index = atomicAdd(reinterpret_cast<uint32_t*>(
			gKernelData.mProfileBuffer), 1) + 1; 

		if(index >= NUM_WARPS_PER_PROFILE_BUFFER)
			return;

		mEvent = reinterpret_cast<warpProfileEvent*>(
			gKernelData.mProfileBuffer) + index;

		fillKernelEvent(*mEvent, 
			gKernelData.mProfileBaseId + id, threadIdx.x );
	}

	__device__ ~ProfileZone() 
	{
		if(mEvent) 
			mEvent->endTime = clock();
	}

	warpProfileEvent* mEvent;
};

#else
typedef ProfileDisabledZone ProfileZone;
#endif

#if 1 // set to 1 to enable detailed profile zones
typedef ProfileZone ProfileDetailZone;
#else
typedef ProfileDisabledZone ProfileDetailZone;
#endif

namespace 
{
	// cut down version of thrust::uninitialized
	// avoids warning about non-empty c'tor
	template<typename T>
	struct uninitialized
	{
		__device__ inline T& get()
		{
			return *reinterpret_cast<T*>(data);
		}

		// maximum alignment required by device code is 16
		__align__(16) unsigned char data[sizeof(T)];
	};
} 

#if __CUDA_ARCH__ < 320
namespace 
{
	__device__ float __ldg(const float* __restrict ptr)
	{
		return *ptr;
	}
}
#endif
   
#define CU_SOLVER_KERNEL_CU               
#include "CuCollision.h"      
#include "CuSelfCollision.h"        

namespace     
{
	__device__ void loadIterData(const cloth::CuIterationData* __restrict iterData)
	{
		if(threadIdx.x < gCuIterationDataSize)
		{
			gIterData.mIntegrationTrafo[threadIdx.x] = 
				iterData->mIntegrationTrafo[threadIdx.x];
		}
	}

	template <typename CurrentT, typename PreviousT>
	__device__ void integrateParticles(CurrentT& current, PreviousT& previous)
	{
		if(gIterData.mIsTurning)
			integrateParticles<true >(current, previous);
		else
			integrateParticles<false>(current, previous);
	}

	// integrate particle positions and store transposed
	template <bool IsTurning, typename CurrentT, typename PreviousT>
	__device__ void integrateParticles(CurrentT& current, PreviousT& previous)
	{
		ProfileDetailZone zone(cloth::CuProfileZoneIds::INTEGRATE);

		const float* __restrict trafo = gIterData.mIntegrationTrafo;

		for(uint32_t i = threadIdx.x; i < gClothData.mNumParticles; i += blockDim.x)
		{
			float nextX = current(i, 0), curX = nextX;
			float nextY = current(i, 1), curY = nextY;
			float nextZ = current(i, 2), curZ = nextZ;			
			float nextW = current(i, 3);

			if(nextW == 0.0f)
				nextW = previous(i, 3);

			if(nextW > 0.0f)
			{
				float prevX = previous(i, 0);
				float prevY = previous(i, 1);
				float prevZ = previous(i, 2);

				if(IsTurning)
				{
					nextX = nextX + trafo[3] +
						curX * trafo[15] + prevX * trafo[ 6] + 
						curY * trafo[16] + prevY * trafo[ 7] + 
						curZ * trafo[17] + prevZ * trafo[ 8];
																								
					nextY = nextY + trafo[4] +
						curX * trafo[18] + prevX * trafo[ 9] +
						curY * trafo[19] + prevY * trafo[10] +
						curZ * trafo[20] + prevZ * trafo[11];

					nextZ = nextZ + trafo[5] +
						curX * trafo[21] + prevX * trafo[12] +
						curY * trafo[22] + prevY * trafo[13] +
						curZ * trafo[23] + prevZ * trafo[14];
				} else {
					nextX += (curX - prevX) * trafo[ 6] + trafo[3];
					nextY += (curY - prevY) * trafo[ 9] + trafo[4];
					nextZ += (curZ - prevZ) * trafo[12] + trafo[5];
				}

				curX  += trafo[0];
				curY  += trafo[1];
				curZ  += trafo[2];
			}

			current(i, 0) = nextX;
			current(i, 1) = nextY;
			current(i, 2) = nextZ;
			current(i, 3) = nextW;

			previous(i, 0) = curX;
			previous(i, 1) = curY;
			previous(i, 2) = curZ;
		}
	}

	template <typename CurrentT>
	__device__ void accelerateParticles(CurrentT& current)
	{
		// might be better to move this into integrate particles
		const float* accelerations = gFrameData.mParticleAccelerations;

		if(!accelerations)
			return;

		ProfileDetailZone zone(cloth::CuProfileZoneIds::ACCELERATE);

		__syncthreads(); // looping with 4 instead of 1 thread per particle

		float sqrIterDt = ~threadIdx.x & 0x3 ? gFrameData.mIterDt * gFrameData.mIterDt : 0.0f;
		typename CurrentT::PointerType sharedCurPos = current[threadIdx.x % 4];

		for(uint32_t i=threadIdx.x; i < gClothData.mNumParticles*4; i += blockDim.x)
		{
			if(current(i/4, 3) > 0.0f)
				sharedCurPos[i/4] += accelerations[i] * sqrIterDt;
		}

		__syncthreads();
	}

	template <typename CurrentT>
	__device__ void constrainTether(CurrentT& current)
	{
		if(0.0f == gFrameData.mTetherConstraintStiffness || !gClothData.mNumTethers)
			return;

		ProfileDetailZone zone(cloth::CuProfileZoneIds::TETHER);

		uint32_t numParticles = gClothData.mNumParticles;
		uint32_t numTethers = gClothData.mNumTethers;
		assert(0 == numTethers % numParticles);

		float stiffness = numParticles * 
			gFrameData.mTetherConstraintStiffness / numTethers;
		float scale = gClothData.mTetherConstraintScale;

		const uint32_t* __restrict tIt = reinterpret_cast<
			const uint32_t*>(gClothData.mTethers);

		for(uint32_t i=threadIdx.x; i<gClothData.mNumParticles; i+=blockDim.x)
		{
			float posX = current(i, 0);
			float posY = current(i, 1);
			float posZ = current(i, 2);

			float offsetX = 0.0f;
			float offsetY = 0.0f;
			float offsetZ = 0.0f;

			for(uint32_t j=i; j<numTethers; j+=gClothData.mNumParticles)
			{
				uint32_t tether = tIt[j];

				uint32_t anchor = tether & 0xffff;
				float deltaX = current(anchor, 0) - posX;
				float deltaY = current(anchor, 1) - posY;
				float deltaZ = current(anchor, 2) - posZ;

				float sqrLength = FLT_EPSILON + deltaX*deltaX 
					+ deltaY*deltaY + deltaZ*deltaZ;

				float radius = (tether >> 16) * scale;
				float slack = 1.0f - radius * rsqrtf(sqrLength);

				if (slack > 0.0f)
				{
					offsetX += deltaX * slack;
					offsetY += deltaY * slack;
					offsetZ += deltaZ * slack;
				}
			}

			current(i, 0) = posX + offsetX * stiffness;
			current(i, 1) = posY + offsetY * stiffness;
			current(i, 2) = posZ + offsetZ * stiffness;
		}

	}

	template <typename CurrentT>
	__device__ void solveFabric(CurrentT& current)
	{
		ProfileDetailZone zone(cloth::CuProfileZoneIds::FABRIC);

		const cloth::CuPhaseConfig* cIt = (cloth::CuPhaseConfig*)gSharedMemory;
		const cloth::CuPhaseConfig* cEnd = cIt + gClothData.mNumPhases;

		for(; cIt != cEnd; ++cIt)
		{
			__syncthreads();

			ProfileDetailZone zone(cloth::CuProfileZoneIds::CONSTRAINT_SET);

			uint32_t numConstraints = cIt->mNumConstaints;
			if(threadIdx.x >= numConstraints)
				continue;

			const float* restvalues = cIt->mRestvalues;
			const float* rIt = restvalues + threadIdx.x;
			const float* rEnd = restvalues + numConstraints;
			const uint16_t* iIt = cIt->mIndices + threadIdx.x * 2;

			float stiffness = cIt->mStiffness;
			float stiffnessMultiplier = cIt->mStiffnessMultiplier;
			float compressionLimit = cIt->mCompressionLimit;
			float stretchLimit = cIt->mStretchLimit;

			uint32_t vpi = iIt[0];
			uint32_t vpj = iIt[1];
			float rij = __ldg(rIt);

			do
			{
				rIt += blockDim.x;
				iIt += blockDim.x * 2;

				uint32_t vpiPrefetch, vpjPrefetch;
				float rijPrefetch;
				if(rIt < rEnd)
				{
					vpiPrefetch = iIt[0];
					vpjPrefetch = iIt[1];
					rijPrefetch = __ldg(rIt);
				}

				float vxi = current(vpi, 0);
				float vyi = current(vpi, 1);
				float vzi = current(vpi, 2);
				float vwi = current(vpi, 3);

				float vxj = current(vpj, 0);
				float vyj = current(vpj, 1);
				float vzj = current(vpj, 2);
				float vwj = current(vpj, 3);

				float hxij = vxj - vxi;
				float hyij = vyj - vyi;
				float hzij = vzj - vzi;

				float e2ij = FLT_EPSILON + hxij*hxij + hyij*hyij + hzij*hzij;
				float negErij = rij > FLT_EPSILON ? -1.0f + rij * rsqrtf(e2ij) : 0.0f;

				negErij = negErij + stiffnessMultiplier * 
					max(compressionLimit, min(-negErij, stretchLimit));

				float negExij = __fdividef(negErij * stiffness, FLT_EPSILON + vwi + vwj);

				float vmi = -vwi * negExij;
				current(vpi, 0) = vxi + vmi * hxij;
				current(vpi, 1) = vyi + vmi * hyij;
				current(vpi, 2) = vzi + vmi * hzij;
				
				float vmj = +vwj * negExij;
				current(vpj, 0) = vxj + vmj * hxij;
				current(vpj, 1) = vyj + vmj * hyij;
				current(vpj, 2) = vzj + vmj * hzij;

				vpi = vpiPrefetch;
				vpj = vpjPrefetch;
				rij = rijPrefetch;

			} while(rIt < rEnd);
		}

		__syncthreads();
	}

#if __CUDA_ARCH__ >= 200 && !defined(_WIN64)
	// ptx version for 32bit shared memory (about 9% faster)
	__device__ void solveFabric(ParticleData<SharedPointer>& current)
	{
		ProfileDetailZone zone(cloth::CuProfileZoneIds::FABRIC);

		const ParticleData<SharedPointer>::PointerType* pointers = current.mPointers;

		asm volatile (
			"{\n\t"
			"	.reg .u32 cIt, cEnd;\n\t"
			"	.reg .pred p;\n\t"
			"	.reg .u32 posPtr, posX, posY, posZ, posW;\n\t"
			"	.reg .u32 tid4, ntid4, numConstraints4;\n\t"
			"	.reg .u32 restvalues, indices;\n\t"
			"	.reg .f32 stiffness, multiplier, compressionLimit, stretchLimit;\n\t"
			"	.reg .u32 rEnd, iIt, rIt;\n\t"
			"	.reg .u32 vpi, vpj;\n\t"
			"	.reg .u32 axj, ayj, azj, awj;\n\t"
			"	.reg .u32 axi, ayi, azi, awi;\n\t"
			"	.reg .f32 vxj, vyj, vzj, vwj;\n\t"
			"	.reg .f32 vxi, vyi, vzi, vwi;\n\t"
			"	.reg .f32 hxij, hyij, hzij;\n\t"
			"	.reg .f32 e2ij;\n\t"
			"	.reg .f32 rsqrtE2ij;\n\t"
			"	.reg .f32 rij;\n\t"
			"	.reg .f32 negErij;\n\t"
			"	.reg .f32 satErij;\n\t"
			"	.reg .f32 negExij;\n\t"
			"	.reg .f32 vwij;\n\t"
			"	.reg .f32 negvwi;\n\t"
			"	.reg .f32 vmi, vmj;\n\t"
			"	ld.shared.u32 cEnd, [gClothData+12];\n\t"
			"	mov.u32 cIt, gSharedMemory;\n\t"
			"   mad.lo.u32 cEnd, cEnd, 28, cIt;\n\t"
			"	setp.eq.u32 p, cIt, cEnd;\n\t"
			"@p bra configEnd;\n\t"
			"	cvta.to.shared.u32 posPtr, %0;\n\t"
			"	ld.shared.u32 posX, [posPtr   ];\n\t"
			"	ld.shared.u32 posY, [posPtr+ 4];\n\t"
			"	ld.shared.u32 posZ, [posPtr+ 8];\n\t"
			"	ld.shared.u32 posW, [posPtr+12];\n\t"
			"	cvta.to.shared.u32 posX, posX;\n\t"
			"	cvta.to.shared.u32 posY, posY;\n\t"
			"	cvta.to.shared.u32 posZ, posZ;\n\t"
			"	cvta.to.shared.u32 posW, posW;\n\t"
			"	mov.u32 tid4, %tid.x;\n\t"
			"	mov.u32 ntid4, %ntid.x;\n\t"
			"	shl.b32 tid4, tid4, 2;\n\t"
			"	shl.b32 ntid4, ntid4, 2;\n\t"
			"configBegin:\n\t"
			"	bar.sync 0;\n\t"
			"	ld.shared.u32 numConstraints4, [cIt+16];\n\t"
			"	shl.b32 numConstraints4, numConstraints4, 2;\n\t"
			"	setp.ge.u32 p, tid4, numConstraints4;\n\t"
			"@p bra constraintEnd;\n\t"
			"	ld.shared.u32 restvalues, [cIt+20];\n\t"
			"	ld.shared.u32 indices, [cIt+24];\n\t"
			"	ld.shared.f32 stiffness, [cIt+0];\n\t"
			"	ld.shared.f32 multiplier, [cIt+4];\n\t"
			"	ld.shared.f32 compressionLimit, [cIt+8];\n\t"
			"	ld.shared.f32 stretchLimit, [cIt+12];\n\t"
			"	add.u32 iIt, indices, tid4;\n\t"
			"	add.u32 rIt, restvalues, tid4;\n\t"
			"	add.u32 rEnd, restvalues, numConstraints4;\n\t"
			"constraintBegin:\n\t"
			"	ld.global.u32 vpi, [iIt];\n\t"
			"	and.b32 vpj, vpi, 0xffff0000;\n\t"
			"	and.b32 vpi, vpi, 0x0000ffff;\n\t"
			"	shr.b32 vpj, vpj, 14;\n\t"
			"	shl.b32 vpi, vpi, 2;\n\t"
			"	add.u32 axj, posX, vpj;\n\t"
			"	add.u32 ayj, posY, vpj;\n\t"
			"	add.u32 azj, posZ, vpj;\n\t"
			"	add.u32 awj, posW, vpj;\n\t"
			"	add.u32 axi, posX, vpi;\n\t"
			"	add.u32 ayi, posY, vpi;\n\t"
			"	add.u32 azi, posZ, vpi;\n\t"
			"	add.u32 awi, posW, vpi;\n\t"
			"	ld.shared.f32 vxj, [axj];\n\t"
			"	ld.shared.f32 vyj, [ayj];\n\t"
			"	ld.shared.f32 vzj, [azj];\n\t"
			"	ld.shared.f32 vwj, [awj];\n\t"
			"	ld.shared.f32 vxi, [axi];\n\t"
			"	ld.shared.f32 vyi, [ayi];\n\t"
			"	ld.shared.f32 vzi, [azi];\n\t"
			"	ld.shared.f32 vwi, [awi];\n\t"
			"	sub.ftz.f32 hxij, vxj, vxi;\n\t"
			"	sub.ftz.f32 hyij, vyj, vyi;\n\t"
			"	sub.ftz.f32 hzij, vzj, vzi;\n\t"
			"	fma.rn.ftz.f32 e2ij, hxij, hxij, 0f34000000;\n\t"
			"	fma.rn.ftz.f32 e2ij, hyij, hyij, e2ij;\n\t"
			"	fma.rn.ftz.f32 e2ij, hzij, hzij, e2ij;\n\t"
			"	rsqrt.approx.ftz.f32 rsqrtE2ij, e2ij;\n\t"
			"	ld.global.f32 rij, [rIt];\n\t"
			"	fma.rn.ftz.f32 negErij, rij, rsqrtE2ij, 0fBF800000;\n\t"
			"	setp.le.ftz.f32 p, rij, 0f34000000;\n\t"
			"@p mov.f32 negErij, 0f00000000;\n\t"
			"	neg.f32 satErij, negErij;\n\t"
			"	min.ftz.f32 satErij, satErij, stretchLimit;\n\t"
			"	max.ftz.f32 satErij, satErij, compressionLimit;\n\t"
			"	fma.rn.ftz.f32 negErij, satErij, multiplier, negErij;\n\t"
			"	mul.ftz.f32 negExij, negErij, stiffness;\n\t"
			"	add.ftz.f32 vwij, vwj, 0f34000000;\n\t"
			"	add.ftz.f32 vwij, vwij, vwi;\n\t"
			"	div.approx.ftz.f32 negExij, negExij, vwij;\n\t"
			"	neg.f32 negvwi, vwi;\n\t"
			"	mul.ftz.f32 vmi, negExij, negvwi;\n\t"
			"	mul.ftz.f32 vmj, negExij, vwj;\n\t"
			"	fma.rn.ftz.f32 vxi, vmi, hxij, vxi;\n\t"
			"	fma.rn.ftz.f32 vyi, vmi, hyij, vyi;\n\t"
			"	fma.rn.ftz.f32 vzi, vmi, hzij, vzi;\n\t"
			"	fma.rn.ftz.f32 vxj, vmj, hxij, vxj;\n\t"
			"	fma.rn.ftz.f32 vyj, vmj, hyij, vyj;\n\t"
			"	fma.rn.ftz.f32 vzj, vmj, hzij, vzj;\n\t"
			"	st.shared.f32 [axj], vxj;\n\t"
			"	st.shared.f32 [ayj], vyj;\n\t"
			"	st.shared.f32 [azj], vzj;\n\t"
			"	st.shared.f32 [axi], vxi;\n\t"
			"	st.shared.f32 [ayi], vyi;\n\t"
			"	st.shared.f32 [azi], vzi;\n\t"
			"	add.u32 rIt, rIt, ntid4;\n\t"
			"	add.u32 iIt, iIt, ntid4;\n\t"
			"	setp.lt.u32 p, rIt, rEnd;\n\t"
			"@p bra constraintBegin;\n\t"
			"constraintEnd:\n\t"
			"	add.u32 cIt, cIt, 28;\n\t"
			"	setp.ne.u32 p, cIt, cEnd;\n\t"
			"@p bra configBegin;\n\t"
			"configEnd:\n\t"
			"	bar.sync 0;\n\t"
			"}"
			: : "r"(pointers) : "memory"
		);
	}
#endif

	template <typename CurrentT>
	__device__ void constrainMotion(CurrentT& current, float alpha)
	{
		if(!gFrameData.mStartMotionConstraints)
			return;

		ProfileDetailZone zone(cloth::CuProfileZoneIds::MOTION);

		// negative because of fused multiply-add optimization
		float negativeScale = -gClothData.mMotionConstraintScale;
		float negativeBias = -gClothData.mMotionConstraintBias;

		for(uint32_t i=threadIdx.x; i<gClothData.mNumParticles; i+=blockDim.x)
		{
			const float* startIt = gFrameData.mStartMotionConstraints + 4*i; // ! bank conflicts
			const float* targetIt = gFrameData.mTargetMotionConstraints + 4*i;

			float sphereX = startIt[0] + (targetIt[0] - startIt[0]) * alpha;
			float sphereY = startIt[1] + (targetIt[1] - startIt[1]) * alpha;
			float sphereZ = startIt[2] + (targetIt[2] - startIt[2]) * alpha;
			float sphereW = startIt[3] + (targetIt[3] - startIt[3]) * alpha;

			float dx = sphereX - current(i, 0);
			float dy = sphereY - current(i, 1);
			float dz = sphereZ - current(i, 2);

			float sqrLength = FLT_EPSILON + dx*dx + dy*dy + dz*dz;
			float negativeRadius = min(0.0f, sphereW * negativeScale + negativeBias);

			float slack = max(negativeRadius * rsqrtf(sqrLength) + 1.0f,
				0.0f) * gFrameData.mMotionConstraintStiffness;

			current(i, 0) += slack * dx;
			current(i, 1) += slack * dy;
			current(i, 2) += slack * dz;

			// set invMass to zero if radius is zero
			if(negativeRadius >= 0.0f)
				current(i, 3) = 0.0f;

		}
	}

	template <typename T>
	__device__ void constrainSeparation(T& current, float alpha)
	{
		if(!gFrameData.mStartSeparationConstraints)
			return;
		
		ProfileDetailZone zone(cloth::CuProfileZoneIds::SEPARATION);

		for(uint32_t i=threadIdx.x; i<gClothData.mNumParticles; i+=blockDim.x)
		{
			const float* startIt = gFrameData.mStartSeparationConstraints + 4*i;
			const float* targetIt = gFrameData.mTargetSeparationConstraints + 4*i;

			float sphereX = startIt[0] + (targetIt[0] - startIt[0]) * alpha;
			float sphereY = startIt[1] + (targetIt[1] - startIt[1]) * alpha;
			float sphereZ = startIt[2] + (targetIt[2] - startIt[2]) * alpha;
			float sphereW = startIt[3] + (targetIt[3] - startIt[3]) * alpha;

			float dx = sphereX - current(i, 0);
			float dy = sphereY - current(i, 1);
			float dz = sphereZ - current(i, 2);

			float sqrLength = FLT_EPSILON + dx*dx + dy*dy + dz*dz;

			float slack = min(0.0f, 1.0f - sphereW * rsqrtf(sqrLength));

			current(i, 0) += slack * dx;
			current(i, 1) += slack * dy;
			current(i, 2) += slack * dz;
		}
	}
 
	template <typename CurrentT, typename PreviousT>
	__device__ void updateSleepState(const CurrentT& current, const PreviousT& previous)
	{
		ProfileDetailZone zone(cloth::CuProfileZoneIds::SLEEP);

		if(!threadIdx.x)
			gFrameData.mSleepTestCounter += max(1, uint32_t(gFrameData.mIterDt*1000));

		__syncthreads();  
 
		if (gFrameData.mSleepTestCounter < gClothData.mSleepTestInterval)
			return;

		float maxDelta = 0.0f;
		for(uint32_t i = threadIdx.x; i < gClothData.mNumParticles; i += blockDim.x)
		{
			maxDelta = max(fabsf(current(i,0) - previous(i,0)), maxDelta);
			maxDelta = max(fabsf(current(i,1) - previous(i,1)), maxDelta);
			maxDelta = max(fabsf(current(i,2) - previous(i,2)), maxDelta);
		}

		if(!threadIdx.x)
		{
			++gFrameData.mSleepPassCounter;
			gFrameData.mSleepTestCounter -= gClothData.mSleepTestInterval;
		}

		__syncthreads();

		if(maxDelta > gClothData.mSleepThreshold*gFrameData.mIterDt)
			gFrameData.mSleepPassCounter = 0;
	}

	template <typename CurrentT, typename PreviousT>
	__device__ void simulateCloth(CurrentT& current, PreviousT& previous)
	{
		// apply exponent to phase configs
		assert(blockDim.x >= gClothData.mNumPhases);
		if(threadIdx.x < gClothData.mNumPhases)
		{
			float exponent = gFrameData.mStiffnessExponent;
			float* ptr = gSharedMemory + threadIdx.x * gCuPhaseConfigSize;
			ptr[0] = 1.0f - exp2f(ptr[0] * exponent);
			ptr[1] = 1.0f - exp2f(ptr[1] * exponent);
		}

		uint32_t numIterations = gFrameData.mNumIterations;
		float invNumIterations = __fdividef(1.0f, numIterations);

		const cloth::CuIterationData* iterData = gFrameData.mIterationData;
		const cloth::CuIterationData* iterEnd = iterData + numIterations;

		loadIterData(iterData);

		__syncthreads();

		for(float alpha = invNumIterations; iterData != iterEnd; alpha += invNumIterations)
		{
			integrateParticles(current, previous);
			accelerateParticles(current);
			constrainMotion(current, alpha);
			constrainTether(current);
			solveFabric(current);
			loadIterData(++iterData);
			constrainSeparation(current, alpha);
			gCollideParticles.get()(current, previous, alpha);
#if __CUDA_ARCH__ < 200 // see DE8360
			if(gFrameData.mRestPositions)
#endif
			gSelfCollideParticles.get()(current);
			updateSleepState(current, previous);
		}

		__syncthreads();
	}

	template <typename CurrentData, typename PreviousData>
	struct ParticleDataPair
	{
		CurrentData mCurrent;
		PreviousData mPrevious;
	};

	typedef ParticleData<SharedPointer> SharedData;
	typedef ParticleData<GlobalPointer> GlobalData;

	__device__ void simulateShared()
	{
		__shared__ uninitialized<ParticleDataPair<SharedData, SharedData>> particles;
		
		uint32_t configDataSize = gClothData.mNumPhases * gCuPhaseConfigSize;
		uint32_t particlesDataSize = 4 * gClothData.mNumParticles;

		SharedPointer<float>::Type sharedCurPos = gSharedMemory +
				configDataSize + threadIdx.x % 4 * gClothData.mNumParticles;
		SharedPointer<float>::Type sharedPrevPos = sharedCurPos + particlesDataSize;

		if(threadIdx.x < 4)
		{
			particles.get().mCurrent.mPointers[threadIdx.x] = sharedCurPos;
			particles.get().mPrevious.mPointers[threadIdx.x] = sharedPrevPos;
		}

		float* globalCurPos = gClothData.mParticles;
		float* globalPrevPos = gClothData.mParticles + particlesDataSize;

		// copy particles from device memory to shared memory and transpose
		for(uint32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
		{
			sharedCurPos[i/4] = globalCurPos[i];
			sharedPrevPos[i/4] = globalPrevPos[i];
		}

		simulateCloth(particles.get().mCurrent, particles.get().mPrevious);

		// copy particles from shared memory to device memory and transpose
		for(uint32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
		{
			globalCurPos[i] = sharedCurPos[i/4];
			globalPrevPos[i] = sharedPrevPos[i/4];
		}

		__syncthreads();
	}

	__device__ void simulateStreamed()
	{
		__shared__ uninitialized<ParticleDataPair<SharedData, GlobalData>> particles;

		uint32_t configDataSize = gClothData.mNumPhases * gCuPhaseConfigSize;
		uint32_t particlesDataSize = 4 * gClothData.mNumParticles;

		float* globalCurPos = gClothData.mParticles;
		SharedPointer<float>::Type sharedCurPos = gSharedMemory +
				configDataSize + threadIdx.x % 4 * gClothData.mNumParticles;

		if(threadIdx.x < 4)
		{
			particles.get().mCurrent.mPointers[threadIdx.x] = sharedCurPos;
			particles.get().mPrevious.mPointers[threadIdx.x] = globalCurPos 
				+ particlesDataSize + threadIdx.x;
		}

		// copy particles from device memory to shared memory and transpose
		for(uint32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
			sharedCurPos[i/4] = globalCurPos[i];

		simulateCloth(particles.get().mCurrent, particles.get().mPrevious);

		// copy particles from shared memory to device memory and transpose
		for(uint32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
			globalCurPos[i] = sharedCurPos[i/4];

		__syncthreads();
	}

	__device__ void simulateGlobal()
	{
		__shared__ uninitialized<ParticleDataPair<GlobalData, GlobalData>> particles;

		if(threadIdx.x < 8)
		{
			particles.get().mCurrent.mPointers[threadIdx.x] = gClothData.mParticles
				+ (threadIdx.x&4)*(gClothData.mNumParticles-1) + threadIdx.x;
		}

		simulateCloth(particles.get().mCurrent, particles.get().mPrevious);
	}

} // anonymous namespace

#if __CUDA_ARCH__ >= 300
__global__ void __launch_bounds__(1024, 1) simulateCloths()
#elif __CUDA_ARCH__ >= 200
__global__ void __launch_bounds__(512, 1) simulateCloths()
#else
__global__ void __launch_bounds__(192, 1) simulateCloths()
#endif
{
	ProfileZone zone(cloth::CuProfileZoneIds::SIMULATE);

	// check that http://nvbugs/1038473 is fixed
	assert(gSharedMemory > (float*)&gFrameData);
	assert(gSharedMemory > (float*)&gClothData);

	// fetch cloth index from queue
	__shared__ uint32_t clothIdx;
	if(!threadIdx.x)
		clothIdx = atomicInc(gKernelData.mClothIndex, gridDim.x-1);
	__syncthreads();
	assert(clothIdx < gridDim.x);

	// copy cloth data to shared memory
	const uint32_t* clothData = reinterpret_cast<const uint32_t*>(gKernelData.mClothData + clothIdx);
	if(threadIdx.x < gCuClothDataSize)
		reinterpret_cast<uint32_t*>(&gClothData)[threadIdx.x] = clothData[threadIdx.x];

	// copy frame data to shared memory
	uint32_t* frameData = reinterpret_cast<uint32_t*>(gKernelData.mFrameData + clothIdx);
	if(threadIdx.x < gCuFrameDataSize)
		reinterpret_cast<uint32_t*>(&gFrameData)[threadIdx.x] = frameData[threadIdx.x];

	__syncthreads();

	if(gFrameData.mSleepPassCounter >= gClothData.mSleepAfterCount)
		return; // cloth is sleeping, exit

	// copy phase configs to shared memory
	uint32_t configDataSize = gClothData.mNumPhases * gCuPhaseConfigSize;
	for(uint32_t i = threadIdx.x; i < configDataSize; i += blockDim.x)
		gSharedUnsigned[i] = reinterpret_cast<const uint32_t*>(gClothData.mPhaseConfigs)[i];

	SharedPointer<uint32_t>::Type scratchPtr = gSharedUnsigned + configDataSize 
		+ 4*gFrameData.mNumSharedPositions*gClothData.mNumParticles;

	// initialize with placement new
	new (gCollideParticles.data) CuCollision(scratchPtr);
	new (gSelfCollideParticles.data) CuSelfCollision();

	// copy particles and constraints to device
	if(gFrameData.mDeviceParticlesDirty)
	{
		for(uint32_t i = threadIdx.x; i < gClothData.mNumParticles*8; i += blockDim.x)
			gClothData.mParticles[i] = gClothData.mParticlesHostCopy[i];
	}
	if(gFrameData.mHostMotionConstraints)
	{
		for(uint32_t i = threadIdx.x; i < gClothData.mNumParticles*4; i += blockDim.x)
			gFrameData.mTargetMotionConstraints[i] = gFrameData.mHostMotionConstraints[i];
	}
	if(gFrameData.mHostSeparationConstraints)
	{
		for(uint32_t i = threadIdx.x; i < gClothData.mNumParticles*4; i += blockDim.x)
			gFrameData.mTargetSeparationConstraints[i] = gFrameData.mHostSeparationConstraints[i];
	}
	if(gFrameData.mHostParticleAccelerations)
	{
		for(uint32_t i = threadIdx.x; i < gClothData.mNumParticles*4; i += blockDim.x)
			gFrameData.mParticleAccelerations[i] = gFrameData.mHostParticleAccelerations[i];
	}

	// necessary to ensure phase configs are fully loaded before setup in simulateCloth()
	__syncthreads();

	switch(gFrameData.mNumSharedPositions) 
	{
	case 0: simulateGlobal(); break; 
	case 1: simulateStreamed(); break;  
	case 2: simulateShared(); break;  
	}
     
	// write back frame data 
	if(threadIdx.x < gCuFrameDataSize)
		frameData[threadIdx.x] = reinterpret_cast<const uint32_t*>(&gFrameData)[threadIdx.x];

	// copy particles to host
	for(uint32_t i = threadIdx.x; i < gClothData.mNumParticles*8; i += blockDim.x)
		gClothData.mParticlesHostCopy[i] = gClothData.mParticles[i];
}

const char* cloth::getKernelDataName()
{
	return "gKernelData";
}

const char* cloth::getKernelFunctionName()
{
	return "_Z14simulateClothsv";
}

