#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.

#include "common.cuh"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::iofx;
#include "include/sortNew.h"

//assuming that the number of warps per block is less or equal to 32
const unsigned int MAX_WARPS_PER_BLOCK = 32;

#ifndef __CUDA_ARCH__
#define __CUDA_ARCH__ 0
#endif

//--------------------------------------------------------------------------------------------------

typedef unsigned int uint;

const uint KEY_BITS_PER_STEP = RADIX_SORT_NBITS;
const uint KEY_DIGITS_PER_STEP = (1U << KEY_BITS_PER_STEP);

#define DEBUG_RADIX_SORT 0
#define KEPLER_BUG_FIX 1

//--------------------------------------------------------------------------------------------------

template <uint VectorSize>
struct VectorTT
{
};

template <>
struct VectorTT<1>
{
	static const uint Log2Size = 0;
	static const uint Size = 1;
	typedef uint AccessType;
};
template <>
struct VectorTT<2>
{
	static const uint Log2Size = 1;
	static const uint Size = 2;
	typedef uint2 AccessType;
};
template <>
struct VectorTT<4>
{
	static const uint Log2Size = 2;
	static const uint Size = 4;
	typedef uint4 AccessType;
};

//--------------------------------------------------------------------------------------------------

template <bool doFlip>
struct KeyTraits
{
};

template <>
struct KeyTraits<false>
{
	static const uint OutOfRangeValue = 0xFFFFFFFFu;

	static inline __device__ uint flip(uint f) { return f; }
	static inline __device__ uint unflip(uint f) { return f; }
};

template <>
struct KeyTraits<true>
{
	static const uint OutOfRangeValue = 0x7FFFFFFFu;

	// ================================================================================================
	// Flip a float for sorting
	//  finds SIGN of fp number.
	//  if it's 1 (negative float), it flips all bits
	//  if it's 0 (positive float), it flips the sign only
	// ================================================================================================
	static inline __device__ uint flip(uint f)
	{
		const uint mask = -int(f >> 31) | 0x80000000;
		return f ^ mask;
	}
	// ================================================================================================
	// flip a float back (invert FloatFlip)
	//  signed was flipped from above, so:
	//  if sign is 1 (negative), it flips the sign bit back
	//  if sign is 0 (positive), it flips all bits back
	// ================================================================================================
	static inline __device__ uint unflip(uint f)
	{
		const uint mask = ((f >> 31) - 1) | 0x80000000;
		return f ^ mask;
	}
};

//--------------------------------------------------------------------------------------------------

#if __CUDA_ARCH__ >= 300

static __device__ __inline__ uint reduceWarp(uint val, volatile uint* sdata)
{
	uint res = val;

	asm volatile (
"{"
"	.reg .u32 tmp;"
"	shfl.bfly.b32 tmp, %0, 0x10, 0x1f;"
"	add.u32 %0, tmp, %0;"
"	shfl.bfly.b32 tmp, %0, 0x08, 0x1f;"
"	add.u32 %0, tmp, %0;"
"	shfl.bfly.b32 tmp, %0, 0x04, 0x1f;"
"	add.u32 %0, tmp, %0;"
"	shfl.bfly.b32 tmp, %0, 0x02, 0x1f;"
"	add.u32 %0, tmp, %0;"
"	shfl.bfly.b32 tmp, %0, 0x01, 0x1f;"
"	add.u32 %0, tmp, %0;"
"}"
	: "+r"(res) : );

	return res;
}

static __device__ __inline__ uint scanWarp(uint val, volatile uint* sdata)
{
	uint ret = val;

	asm volatile (
"{"
"	.reg .u32 tmp;"
"	.reg .pred p;"
"	shfl.up.b32 tmp|p, %0, 0x1, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x2, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x4, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x8, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x10, 0x0;"
"@p add.u32 %0, tmp, %0;"
"}"
	: "+r"(ret) : );

	return ret;
}

static __device__ __inline__ uint scanWarpWithTotal(uint val, uint& total, volatile uint* sdata)
{
	uint ret = val;
	uint out;

	asm volatile (
"{	.reg .u32 tmp;"
"	.reg .pred p;"
"	shfl.up.b32 tmp|p, %0, 0x1, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x2, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x4, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x8, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.up.b32 tmp|p, %0, 0x10, 0x0;"
"@p add.u32 %0, tmp, %0;"
"	shfl.idx.b32 %1, %0, 0x1f, 0x1f;"
"}"
	: "+r"(ret), "=r"(out) : );

	total = out;
	return ret;
}

static __device__ __inline__ uint SHFL(uint val, uint idx)
{
	uint res;

	asm volatile (
"	shfl.idx.b32 %0, %1, %2, 0x1f;"
	: "=r"(res) : "r"(val), "r"(idx) );

	return res;
}

#else

static __device__ __inline__ uint reduceWarp(uint val, volatile uint* sdata)
{
	unsigned int idx = threadIdx.x;
	sdata[idx] = val;
	if ((idx & (WARP_SIZE-1)) < 16)
	{
		sdata[idx] = sdata[idx] + sdata[idx + 16]; 
		sdata[idx] = sdata[idx] + sdata[idx +  8];
		sdata[idx] = sdata[idx] + sdata[idx +  4];
		sdata[idx] = sdata[idx] + sdata[idx +  2];
		sdata[idx] = sdata[idx] + sdata[idx +  1];
	}
	unsigned int res = sdata[idx & ~(WARP_SIZE-1)];
	return res;
}

static __device__ __inline__ uint scanWarp(uint val, volatile uint* sdata)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);

	//setup scan
	uint scanIdx = (warpIdx << (LOG2_WARP_SIZE + 1)) + idxInWarp;
	sdata[scanIdx] = 0;
	scanIdx += WARP_SIZE;
	sdata[scanIdx] = val;

	sdata[scanIdx] += sdata[scanIdx -  1]; 
	sdata[scanIdx] += sdata[scanIdx -  2]; 
	sdata[scanIdx] += sdata[scanIdx -  4]; 
	sdata[scanIdx] += sdata[scanIdx -  8]; 
	sdata[scanIdx] += sdata[scanIdx - 16];

	return sdata[scanIdx];
}

static __device__ __inline__ uint scanWarpWithTotal(uint val, uint& total, volatile uint* sdata)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);

	//setup scan
	uint scanIdx = (warpIdx << (LOG2_WARP_SIZE + 1)) + idxInWarp;
	sdata[scanIdx] = 0;
	scanIdx += WARP_SIZE;
	sdata[scanIdx] = val;

	sdata[scanIdx] += sdata[scanIdx -  1]; 
	sdata[scanIdx] += sdata[scanIdx -  2]; 
	sdata[scanIdx] += sdata[scanIdx -  4]; 
	sdata[scanIdx] += sdata[scanIdx -  8]; 
	sdata[scanIdx] += sdata[scanIdx - 16];

	total = sdata[scanIdx | (WARP_SIZE-1)];
	return sdata[scanIdx];
}

#endif

//--------------------------------------------------------------------------------------------------

inline __device__ void scanBlockInSMem(uint scanCount, uint* scanArray, volatile uint* sdata)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);

	const uint scanWarps = (scanCount + WARP_SIZE-1) >> LOG2_WARP_SIZE;

	uint scanRes;

	__shared__ volatile uint sScanForWarp[MAX_WARPS_PER_BLOCK];
	if (warpIdx < scanWarps)
	{
		uint scanVal = (idx < scanCount) ? scanArray[idx] : 0;
		scanRes = scanWarp(scanVal, sdata);

		if (idxInWarp == WARP_SIZE-1)
		{
			sScanForWarp[warpIdx] = scanRes;
		}
		scanRes -= scanVal; //make scan exclusive
	}
	__syncthreads();

	//1 warp scan
	if (idx < WARP_SIZE)
	{
		uint warpScanVal = (idx < scanWarps) ? sScanForWarp[idx] : 0;
		uint warpScanRes = scanWarp(warpScanVal, sdata);

		if (idxInWarp == WARP_SIZE-1)
		{
			//store total scan at the end of scanArray
			scanArray[scanCount] = warpScanRes;
		}
		warpScanRes -= warpScanVal; //make scan exclusive
		sScanForWarp[idx] = warpScanRes; 
	}
	__syncthreads();

	if (idx < scanCount)
	{
		scanArray[idx] = scanRes + sScanForWarp[warpIdx];
	}
}

inline __device__ uint4 scan4(uint scanCount, uint4 idata, volatile uint* sData, uint* pOutTotal = 0)
{    
	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);

	const uint scanWarps = (scanCount + WARP_SIZE-1) >> LOG2_WARP_SIZE;

	__shared__ volatile uint sScanForWarp[MAX_WARPS_PER_BLOCK];

	uint4 val4 = idata;
	uint res;
	uint sum[3];
	if (warpIdx < scanWarps)
	{
		sum[0] = val4.x;
		sum[1] = val4.y + sum[0];
		sum[2] = val4.z + sum[1];

		uint val = val4.w + sum[2];
		res = scanWarp(val, sData);

		if (idxInWarp == WARP_SIZE-1)
		{
			sScanForWarp[warpIdx] = res;
		}
		res -= val; //make scan exclusive
	}
	__syncthreads();

	//1 warp scan
	if (idx < WARP_SIZE)
	{
		uint warpScanVal = (idx < scanWarps) ? sScanForWarp[idx] : 0;
		uint warpScanRes = scanWarp(warpScanVal, sData);

		if (idxInWarp == WARP_SIZE-1)
		{
			if (pOutTotal != 0)
			{
				*pOutTotal = warpScanRes;
			}
		}
		warpScanRes -= warpScanVal; //make scan exclusive
		sScanForWarp[idx] = warpScanRes; 
	}
	__syncthreads();

	if (warpIdx < scanWarps)
	{
		res += sScanForWarp[warpIdx];
		val4.x = res;
		val4.y = res + sum[0];
		val4.z = res + sum[1];
		val4.w = res + sum[2];
	}
	return val4;
}

//--------------------------------------------------------------------------------------------------

template <uint WarpsPerBlock, uint VectorSize, uint WarpStride, bool FullBlock>
inline __device__ void localSortStep(uint count, uint startBit, uint key[VectorSize], uint value[VectorSize], 
							  uint* sKeys, uint* sValues, uint* sWarpCounters, volatile uint* sdata,
							  uint remainWarps)
{
	const unsigned int ElemsPerThread = VectorTT<VectorSize>::Size;
	typedef typename VectorTT<VectorSize>::AccessType ElemAccessType;

	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);

	if (!(FullBlock || warpIdx < remainWarps))
	{
		//fill gaps for unused warps in sWarpCounters with 0 for correct scan later
		if (idxInWarp < KEY_DIGITS_PER_STEP)
		{
			sWarpCounters[idxInWarp * WarpStride + warpIdx] = 0;
		}
	}
	//fill gaps out of warps in sWarpCounters with 0 for correct scan later
	if (WarpStride > WarpsPerBlock && idx < KEY_DIGITS_PER_STEP)
	{
		sWarpCounters[idx * WarpStride + WarpsPerBlock] = 0;
	}

	uint keyDigit[ElemsPerThread];
	uint keyOffset[ElemsPerThread];

	if (FullBlock || warpIdx < remainWarps)
	{
		#pragma unroll
		for (int i = 0; i < ElemsPerThread; ++i)
		{
			keyDigit[i] = ((key[i] >> startBit) & (KEY_DIGITS_PER_STEP - 1));
			keyOffset[i] = 0;
		}

		#pragma unroll
		for (uint bit = 0; bit < KEY_DIGITS_PER_STEP; bit += 4)
		{
			//seq. reduce
			uint scanVal = 0;
			#pragma unroll
			for (int i = 0; i < ElemsPerThread; ++i)
			{
				scanVal += (1u << ((keyDigit[i] - bit) << 3)); //in PTX shifts are clamped to 32, so it's ok here
			}

			uint scanTotal;
			uint scanRes = scanWarpWithTotal(scanVal, scanTotal, sdata);

			if (idxInWarp < 4)
			{
				sWarpCounters[(bit + idxInWarp)*WarpStride + warpIdx] = ((scanTotal >> (idxInWarp << 3)) & 0xFF);
			}
			scanRes -= scanVal; //makes scan exclusive

			#pragma unroll
			for (int i = 0; i < ElemsPerThread; ++i)
			{
				keyOffset[i] |= ((scanRes >> ((keyDigit[i] - bit) << 3)) & 0xFF); //in PTX shifts are clamped to 32, so it's ok here
			}
		}
	}
	__syncthreads();

	const uint scanCount = KEY_DIGITS_PER_STEP * WarpStride;
	scanBlockInSMem(scanCount, sWarpCounters, sdata);
	__syncthreads();

	if (FullBlock || warpIdx < remainWarps)
	{
#if (__CUDA_ARCH__ >= 300)
		uint keyOffsetForWarp;
		if (idxInWarp < KEY_DIGITS_PER_STEP) keyOffsetForWarp = sWarpCounters[idxInWarp*WarpStride + warpIdx];
#endif
		//seq. exclusive scan
		const uint AccumCount = (KEY_DIGITS_PER_STEP / 16);
		uint accum[AccumCount];
		#pragma unroll
		for (int k = 0; k < AccumCount; ++k) accum[k] = 0;

		#pragma unroll
		for (int i = 0; i < ElemsPerThread; ++i)
		{
			uint digit = keyDigit[i];

			uint keyLocalOffset = keyOffset[i];
			#pragma unroll
			for (int k = 0; k < AccumCount; ++k) keyLocalOffset += ((accum[k] >> ((digit - k*16) << 1)) & 3);

#if (__CUDA_ARCH__ >= 300)
			keyLocalOffset += SHFL(keyOffsetForWarp, digit);
#else
			keyLocalOffset += sWarpCounters[digit*WarpStride + warpIdx];
#endif

			sKeys[keyLocalOffset] = key[i];
#if !DEBUG_RADIX_SORT
			sValues[keyLocalOffset] = value[i];
#else
			sValues[idx*ElemsPerThread + i] = keyLocalOffset;
#endif

			#pragma unroll
			for (int k = 0; k < AccumCount; ++k) accum[k] += (1u << ((digit - k*16) << 1));
		}
	}
}

template <uint VectorSize, bool doFlip, bool FullBlock>
inline __device__ void readKeyAndValue(uint count, uint* inpKeys, uint* inpValues, uint key[VectorSize], uint value[VectorSize], uint remainWarps, uint blockPos)
{
	const unsigned int Log2ElemsPerThread = VectorTT<VectorSize>::Log2Size;
	const unsigned int ElemsPerThread = VectorTT<VectorSize>::Size;
	typedef typename VectorTT<VectorSize>::AccessType ElemAccessType;

	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);

	if (FullBlock || warpIdx < remainWarps)
	{
		uint inpPos = blockPos + idx;
		if (FullBlock || inpPos < (count >> Log2ElemsPerThread))
		{
			*((ElemAccessType*)key) = ((ElemAccessType*)inpKeys)[inpPos];
			*((ElemAccessType*)value) = ((ElemAccessType*)inpValues)[inpPos];
		}
		else
		{
			if (VectorSize == 1)
			{
				key[0] = KeyTraits<doFlip>::OutOfRangeValue;
			}
			else
			{
				inpPos <<= Log2ElemsPerThread;
				#pragma unroll
				for (int i = 0; i < ElemsPerThread; ++i, ++inpPos)
				{
					key[i] = KeyTraits<doFlip>::OutOfRangeValue;
					if (inpPos < count)
					{
						key[i] = inpKeys[inpPos];
						value[i] = inpValues[inpPos];
					}
				}
			}
		}

		if (doFlip)
		{
			#pragma unroll
			for (int i = 0; i < ElemsPerThread; ++i)
			{
				key[i] = KeyTraits<true>::flip(key[i]);
			}
		}
	}
}

template <uint WarpsPerBlock, uint VectorSize, bool doFlip, uint WarpStride, bool FullBlock>
inline __device__ void localSortStepBlock(uint count, uint startBit, uint* inpKeys, uint* inpValues, uint* outKeys, uint* outValues,
							  uint* sKeys, uint* sValues, uint* sWarpCounters, volatile uint* sdata, uint* sCounters,
							  uint remainDataCount, uint blockPos)
{
	const unsigned int Log2ElemsPerThread = VectorTT<VectorSize>::Log2Size;
	const unsigned int ElemsPerThread = VectorTT<VectorSize>::Size;
	typedef typename VectorTT<VectorSize>::AccessType ElemAccessType;

	const unsigned int idx = threadIdx.x;

	const unsigned int Log2DataWarpSize = (LOG2_WARP_SIZE + Log2ElemsPerThread);
	const unsigned int DataWarpSize = (1 << Log2DataWarpSize);

	uint remainWarps = (remainDataCount + (DataWarpSize-1)) >> Log2DataWarpSize;

	uint key[ElemsPerThread];
	uint value[ElemsPerThread];

	readKeyAndValue<VectorSize, doFlip, FullBlock>(count, inpKeys, inpValues, key, value, remainWarps, blockPos);

	localSortStep<WarpsPerBlock, VectorSize, WarpStride, FullBlock>(count, startBit, key, value, sKeys, sValues, sWarpCounters, sdata, remainWarps);

	__shared__ int sGlobalOffsets[KEY_DIGITS_PER_STEP];
	if (idx < KEY_DIGITS_PER_STEP)
	{
		uint radixStart = sWarpCounters[idx*WarpStride];
		uint radixEnd = sWarpCounters[(idx+1)*WarpStride];

		sGlobalOffsets[idx] = sCounters[idx] - radixStart;
		sCounters[idx] += (radixEnd - radixStart);
	}
	__syncthreads();

//#if __CUDA_ARCH__ >= 200
#if 1

#if (__CUDA_ARCH__ >= 300) && !KEPLER_BUG_FIX
	int globalOffsetForWarp = 0;
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);
	if (idxInWarp < KEY_DIGITS_PER_STEP) globalOffsetForWarp = sGlobalOffsets[idxInWarp];
#endif
	const unsigned int BlockSize = (WarpsPerBlock << LOG2_WARP_SIZE);
	for (int outIdx = idx; outIdx < remainDataCount; outIdx += BlockSize)
	{
		uint _key = sKeys[outIdx];
		uint _value = sValues[outIdx];
		uint digit = ((_key >> startBit) & (KEY_DIGITS_PER_STEP - 1));
#if (__CUDA_ARCH__ >= 300) && !KEPLER_BUG_FIX
		int globalOffset = SHFL(globalOffsetForWarp, digit);
#else
		int globalOffset = sGlobalOffsets[digit];
#endif

		uint outPos = globalOffset + outIdx;
#if DEBUG_RADIX_SORT
		if (outPos >= 0 && outPos < count)
#endif
		outKeys[outPos] = KeyTraits<doFlip>::unflip(_key);
#if !DEBUG_RADIX_SORT
		outValues[outPos] = _value;
#else
		outValues[blockPos*ElemsPerThread + outIdx] = globalOffset;//_value;
#endif
	}

#if 0
	//manual coalescing full-warp
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	for (uint digit = warpIdx; digit < KEY_DIGITS_PER_STEP; digit += WarpsPerBlock)
	{
		const uint warpOffset = idx & (WARP_SIZE-1);

		const uint radixStart = sWarpCounters[digit * WarpStride];
		const uint radixEnd = sWarpCounters[(digit + 1) * WarpStride];

		const uint endPos = sCounters[digit];
		const uint startPos = endPos - (radixEnd - radixStart);

		const uint leadingInvalid = startPos & (WARP_SIZE-1);

		uint outOffset = (startPos & ~(WARP_SIZE-1)) + warpOffset;
		if (warpOffset >= leadingInvalid && outOffset < endPos)
		{
			uint inOffset0 = radixStart + (warpOffset - leadingInvalid);
			outKeys[outOffset]   = KeyTraits<doFlip>::unflip(sKeys[inOffset0]);
			outValues[outOffset] = sValues[inOffset0];
		}

		outOffset += WARP_SIZE;
		uint inOffset  = radixStart + (WARP_SIZE - leadingInvalid) + warpOffset;

		for (; outOffset < endPos; outOffset += WARP_SIZE, inOffset += WARP_SIZE)
		{
			outKeys[outOffset]   = KeyTraits<doFlip>::unflip(sKeys[inOffset]);
			outValues[outOffset] = sValues[inOffset];
		}
	}
#endif

#else
	//manual coalescing half-warp for Tesla
	const uint halfWarpIdx = (idx >> 4);
	for (uint digit = halfWarpIdx; digit < KEY_DIGITS_PER_STEP; digit += WarpsPerBlock)
	{
		const uint halfWarpOffset = (idx & 0xF);

		const uint radixStart = sWarpCounters[digit * WarpStride];
		const uint radixEnd = sWarpCounters[(digit + 1) * WarpStride];

		const uint endPos = sCounters[digit];
		const uint startPos = endPos - (radixEnd - radixStart);

		const uint leadingInvalid = (startPos & 0xF);

		uint outOffset = (startPos & ~0xF) + halfWarpOffset;
		if (halfWarpOffset >= leadingInvalid && outOffset < endPos)
		{
			uint inOffset0 = radixStart + (halfWarpOffset - leadingInvalid);
			outKeys[outOffset]   = KeyTraits<doFlip>::unflip(sKeys[inOffset0]);
			outValues[outOffset] = sValues[inOffset0];
		}

		outOffset += 16;
		uint inOffset  = radixStart + (16 - leadingInvalid) + halfWarpOffset;

		for (; outOffset < endPos; outOffset += 16, inOffset += 16)
		{
			outKeys[outOffset]   = KeyTraits<doFlip>::unflip(sKeys[inOffset]);
			outValues[outOffset] = sValues[inOffset];
		}
	}
#endif
}

template <uint WarpStride>
inline __device__ void processReduce(uint accum[KEY_DIGITS_PER_STEP >> 2], volatile uint* sData, volatile uint* sWarpCounters)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);

	#pragma unroll
	for (uint keyDigit = 0; keyDigit < KEY_DIGITS_PER_STEP; ++keyDigit)
	{
		uint val = (accum[keyDigit >> 2] >> ((keyDigit & 3) << 3)) & 0xFF;

		uint res = reduceWarp(val, sData);

		if (idxInWarp == 0)
		{
			sWarpCounters[keyDigit*WarpStride + warpIdx] += res;
		}
	}
}

template <uint WarpsPerBlock, uint VectorSize, bool doFlip>
inline __device__ void radixSortStep1(uint count, uint startBit, uint* inpKeys, uint* inpValues, uint* outKeys, uint* outValues, uint* tempScan)
{
	const unsigned int Log2ElemsPerThread = VectorTT<VectorSize>::Log2Size;
	const unsigned int ElemsPerThread = VectorTT<VectorSize>::Size;
	typedef typename VectorTT<VectorSize>::AccessType ElemAccesType;

	const unsigned int Log2DataWarpSize = (LOG2_WARP_SIZE + Log2ElemsPerThread);
	const unsigned int DataWarpSize = (1 << Log2DataWarpSize);

	const unsigned int GridDataWarpsCount = ((count + DataWarpSize-1) >> Log2DataWarpSize);
	const unsigned int DataWarpsResidue = (GridDataWarpsCount % gridDim.x);
	const unsigned int DataWarpsExtra = (blockIdx.x < DataWarpsResidue) ? 1 : 0;
	const unsigned int DataWarpsCount = (GridDataWarpsCount / gridDim.x) + DataWarpsExtra;
	const unsigned int DataWarpsOffset = blockIdx.x * DataWarpsCount + DataWarpsResidue * (1 - DataWarpsExtra);

	const unsigned int BlockSize = (WarpsPerBlock << LOG2_WARP_SIZE);

	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);


	//temp shared memory for scan
	__shared__ volatile uint sdata[BlockSize*2];

	const uint WarpStride = WarpsPerBlock + 1;
	__shared__ volatile uint sWarpCounters[KEY_DIGITS_PER_STEP * WarpStride];

	if (idxInWarp < KEY_DIGITS_PER_STEP)
	{
		sWarpCounters[idxInWarp*WarpStride + warpIdx] = 0;
	}

	uint accum[KEY_DIGITS_PER_STEP >> 2];
	uint accumCount = 0;
	#pragma unroll
	for (uint i = 0; i < (KEY_DIGITS_PER_STEP >> 2); ++i)
	{
		accum[i] = 0;
	}

	uint blockPos = (DataWarpsOffset << LOG2_WARP_SIZE);
	for (int remainWarps = DataWarpsCount; remainWarps > 0; remainWarps -= WarpsPerBlock, blockPos += BlockSize)
	{
		uint key[ElemsPerThread];

		if (warpIdx < remainWarps)
		{
			uint inpPos = blockPos + idx;
			if (inpPos < (count >> Log2ElemsPerThread))
			{
				*((ElemAccesType*)key) = ((ElemAccesType*)inpKeys)[inpPos];
			}
			else
			{
				inpPos <<= Log2ElemsPerThread;
				#pragma unroll
				for (int i = 0; i < ElemsPerThread; ++i, ++inpPos)
				{
					key[i] = KeyTraits<doFlip>::OutOfRangeValue;
					if (inpPos < count)
					{
						key[i] = inpKeys[inpPos];
					}
				}
			}

			#pragma unroll
			for (int i = 0; i < ElemsPerThread; ++i)
			{
				uint keyDigit = ((KeyTraits<doFlip>::flip(key[i]) >> startBit) & (KEY_DIGITS_PER_STEP - 1));

				#pragma unroll
				for (uint bit = 0; bit < KEY_DIGITS_PER_STEP; bit += 4)
				{
					accum[bit >> 2] += (1u << ((keyDigit - bit) << 3)); //in PTX shifts are clamped to 32, so it's ok here
				}
			}

			accumCount += ElemsPerThread;
			//check overflow
			if (accumCount + ElemsPerThread > 0xFF)
			{
				processReduce<WarpStride>(accum, sdata, sWarpCounters);

				accumCount = 0;
				#pragma unroll
				for (uint i = 0; i < (KEY_DIGITS_PER_STEP >> 2); ++i)
				{
					accum[i] = 0;
				}
			}
		}
	}
	if (accumCount > 0)
	{
		processReduce<WarpStride>(accum, sdata, sWarpCounters);
	}
	__syncthreads();

	#pragma unroll
	for (uint digit = warpIdx; digit < KEY_DIGITS_PER_STEP; digit += WarpsPerBlock)
	{
		uint val = (idxInWarp < WarpsPerBlock) ? sWarpCounters[digit*WarpStride + idxInWarp] : 0;
		uint res = reduceWarp(val, sdata);

		if (idxInWarp == 0)
		{
			tempScan[gridDim.x*digit + blockIdx.x] = res;
		}
	}
}


template <uint WarpsPerBlock, uint VectorSize, bool doFlip>
inline __device__ void radixSortStep2(int count, int blockCount, uint* tempScan)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int ScanCount = ((blockCount * KEY_DIGITS_PER_STEP) >> 2);

	const unsigned int BlockSize = (WarpsPerBlock << LOG2_WARP_SIZE);
	__shared__ volatile uint sdata[BlockSize*2];

	uint4 val = (idx < ScanCount) ? ((uint4*)tempScan)[idx] : make_uint4(0, 0, 0, 0);

	val = scan4(ScanCount, val, sdata);

	if (idx < ScanCount)
	{
		val.x = min(val.x, count);
		val.y = min(val.y, count);
		val.z = min(val.z, count);
		val.w = min(val.w, count);
		((uint4*)tempScan)[idx] = val;
	}
}

template <uint WarpsPerBlock, uint VectorSize, bool doFlip>
inline __device__ void radixSortStep3(uint count, uint startBit, uint* inpKeys, uint* inpValues, uint* outKeys, uint* outValues, uint* tempScan)
{
	const unsigned int Log2ElemsPerThread = VectorTT<VectorSize>::Log2Size;
	typedef typename VectorTT<VectorSize>::AccessType ElemAccessType;

	const unsigned int Log2DataWarpSize = (LOG2_WARP_SIZE + Log2ElemsPerThread);
	const unsigned int DataWarpSize = (1 << Log2DataWarpSize);

	const unsigned int GridDataWarpsCount = ((count + DataWarpSize-1) >> Log2DataWarpSize);
	const unsigned int DataWarpsResidue = (GridDataWarpsCount % gridDim.x);
	const unsigned int DataWarpsExtra = (blockIdx.x < DataWarpsResidue) ? 1 : 0;
	const unsigned int DataWarpsCount = (GridDataWarpsCount / gridDim.x) + DataWarpsExtra;
	const unsigned int DataWarpsOffset = blockIdx.x * DataWarpsCount + DataWarpsResidue * (1 - DataWarpsExtra);

	if (DataWarpsCount == 0)
	{
		return;
	}

	const unsigned int BlockSize = (WarpsPerBlock << LOG2_WARP_SIZE);
	const unsigned int BlockDataSize = (WarpsPerBlock << Log2DataWarpSize);

	const unsigned int idx = threadIdx.x;

	const uint WarpStride = WarpsPerBlock + 1; //+1 here is to avoid shared memory bank conflicts
	__shared__ uint sWarpCounters[KEY_DIGITS_PER_STEP * WarpStride + 1]; //+1 here is to store total scan at the end

	__shared__ uint sCounters[KEY_DIGITS_PER_STEP];
	if (idx < KEY_DIGITS_PER_STEP)
	{
		sCounters[idx] = tempScan[gridDim.x*idx + blockIdx.x];
	}

	//temp shared memory for scan
	__shared__ volatile uint sdata[BlockSize*2];

	__shared__ uint sKeys[BlockDataSize];
	__shared__ uint sValues[BlockDataSize];

#if DEBUG_RADIX_SORT
	for (int i = idx; i < BlockDataSize; i += BlockSize)
	{
		sKeys[i] = -2;
		sValues[i] = -2;
	}
	__syncthreads();
#endif


	uint blockBeg = (DataWarpsOffset << Log2DataWarpSize);
	uint blockEnd = min(blockBeg + (DataWarpsCount << Log2DataWarpSize), count);
	int remainDataCount = (blockEnd - blockBeg);

	uint blockPos = (DataWarpsOffset << LOG2_WARP_SIZE);

	for (; remainDataCount >= BlockDataSize; remainDataCount -= BlockDataSize, blockPos += BlockSize)
	{
		localSortStepBlock<WarpsPerBlock, VectorSize, doFlip, WarpStride, true>(
			count, startBit, inpKeys, inpValues, outKeys, outValues, 
			sKeys, sValues, sWarpCounters, sdata, sCounters,
			BlockDataSize, blockPos);
	}
	if (remainDataCount > 0)
	{
		localSortStepBlock<WarpsPerBlock, VectorSize, doFlip, WarpStride, false>(
			count, startBit, inpKeys, inpValues, outKeys, outValues, 
			sKeys, sValues, sWarpCounters, sdata, sCounters,
			remainDataCount, blockPos);
	}
}


template <uint WarpsPerBlock, uint VectorSize, bool doFlip>
inline __device__ void radixSortBlock(uint count, uint startBit0, uint startBit1, uint* inpKeys, uint* inpValues)
{
	const unsigned int Log2ElemsPerThread = VectorTT<VectorSize>::Log2Size;
	const unsigned int ElemsPerThread = VectorTT<VectorSize>::Size;
	typedef typename VectorTT<VectorSize>::AccessType ElemAccessType;

	const unsigned int Log2DataWarpSize = (LOG2_WARP_SIZE + Log2ElemsPerThread);
	const unsigned int DataWarpSize = (1 << Log2DataWarpSize);

	const unsigned int DataWarpsCount = ((count + DataWarpSize-1) >> Log2DataWarpSize);

	const unsigned int BlockSize = (WarpsPerBlock << LOG2_WARP_SIZE);
	const unsigned int BlockDataSize = (WarpsPerBlock << Log2DataWarpSize);

	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);

	const uint WarpStride = WarpsPerBlock + 1; //+1 here is to avoid shared memory bank conflicts
	__shared__ uint sWarpCounters[KEY_DIGITS_PER_STEP * WarpStride + 1]; //+1 here is to store total scan at the end

	//temp shared memory for scan
	__shared__ volatile uint sdata[BlockSize*2];

	__shared__ uint sKeys[BlockDataSize];
	__shared__ uint sValues[BlockDataSize];

	//read from memory
	uint key[ElemsPerThread];
	uint value[ElemsPerThread];

	readKeyAndValue<VectorSize, doFlip, false>(count, inpKeys, inpValues, key, value, DataWarpsCount, 0);

	for (uint startBit = startBit0; startBit < startBit1; startBit += KEY_BITS_PER_STEP)
	{
		localSortStep<WarpsPerBlock, VectorSize, WarpStride, false>(count, startBit, key, value, sKeys, sValues, sWarpCounters, sdata, DataWarpsCount);

		__syncthreads();

		if (warpIdx < DataWarpsCount)
		{
			#pragma unroll
			for (int i = 0; i < ElemsPerThread; ++i)
			{
				key[i] = sKeys[(idx << Log2ElemsPerThread) + i];
				value[i] = sValues[(idx << Log2ElemsPerThread) + i];
			}
		}
	}

	//output to memory
	if (warpIdx < DataWarpsCount)
	{
		if (doFlip)
		{
			#pragma unroll
			for (int i = 0; i < ElemsPerThread; ++i)
			{
				key[i] = KeyTraits<true>::unflip(key[i]);
			}
		}

		uint inpPos = idx;
		if (inpPos < (count >> Log2ElemsPerThread))
		{
			((ElemAccessType*)inpKeys)[inpPos] = *((ElemAccessType*)key);
			((ElemAccessType*)inpValues)[inpPos] = *((ElemAccessType*)value);
		}
		else
		{
			inpPos <<= Log2ElemsPerThread;
			#pragma unroll
			for (int i = 0; i < ElemsPerThread; ++i, ++inpPos)
			{
				if (inpPos < count)
				{
					inpKeys[inpPos] = key[i];
					inpValues[inpPos] = value[i];
				}
			}
		}
	}
}

//--------------------------------------------------------------------------------------------------

BOUND_KERNEL_BEG(SORT_NEW_WARPS_PER_BLOCK, newRadixSortBlockKernel,
	uint count, uint bitCount, uint startBit, uint* inpKeys, uint* inpValues
)
	radixSortBlock<WarpsPerBlock, SORT_NEW_VECTOR_SIZE, false>(count, startBit, startBit + bitCount, inpKeys, inpValues);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SORT_NEW_WARPS_PER_BLOCK, newRadixSortStep1Kernel,
	uint count, uint startBit, uint* inpKeys, uint* inpValues, uint* outKeys, uint* outValues, uint* tempScan
)
	radixSortStep1<WarpsPerBlock, SORT_NEW_VECTOR_SIZE, false>(count, startBit, inpKeys, inpValues, outKeys, outValues, tempScan);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SORT_NEW_WARPS_PER_BLOCK, newRadixSortStep2Kernel,
	uint count, uint blockCount, uint* tempScan
)
	radixSortStep2<WarpsPerBlock, SORT_NEW_VECTOR_SIZE, false>(count, blockCount, tempScan);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SORT_NEW_WARPS_PER_BLOCK, newRadixSortStep3Kernel,
	uint count, uint startBit, uint* inpKeys, uint* inpValues, uint* outKeys, uint* outValues, uint* tempScan
)
	radixSortStep3<WarpsPerBlock, SORT_NEW_VECTOR_SIZE, false>(count, startBit, inpKeys, inpValues, outKeys, outValues, tempScan);

BOUND_KERNEL_END()
