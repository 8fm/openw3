#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.


#include "common.cuh"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::iofx;
#include "include/remap.h"

__device__
PX_INLINE unsigned int floatFlip(float f)
{
    unsigned int i = __float_as_int(f);
	unsigned int mask = -int(i >> 31) | 0x80000000;
	return i ^ mask;
}

__device__
PX_INLINE physx::PxU32 getActorIndex(physx::PxU32 inputID, const NiIofxActorID* actorID, physx::PxU32 numActorClasses, physx::PxU32 numActorIDs)
{
	NiIofxActorID id;
	id.value = tex1Dfetch(KERNEL_TEX_REF(RemapActorIDs), inputID);

	return (id.getVolumeID() == NiIofxActorID::NO_VOLUME) ? (numActorIDs << NiIofxActorID::ASSETS_PER_MATERIAL_BITS)
		: numActorClasses * id.getVolumeID() + id.getActorClassID();
}

/* State Space */
BOUND_KERNEL_BEG( REMAP_WARPS_PER_BLOCK, makeSortKeys,
	const physx::PxU32* inStateToInput, physx::PxU32 maxInputID,
	const NiIofxActorID* actorID, physx::PxU32 numActorClasses, physx::PxU32 numActorIDs,
	const float4* positionMass, bool outputDensityKeys,
	physx::PxVec3 eyePos, physx::PxVec3 eyeDir, physx::PxF32 zNear,
	physx::PxU32* sortKey, physx::PxU32* sortValue
)
	const physx::PxU32 maxStateID = _threadCount;
	for (physx::PxU32 stateID = BlockSize*blockIdx.x + threadIdx.x; stateID < maxStateID; stateID += BlockSize*gridDim.x)
	{
		physx::PxU32 key = outputDensityKeys ? 0xFFFFFFFFu : ((numActorIDs + 1) << NiIofxActorID::ASSETS_PER_MATERIAL_BITS);
		physx::PxU32 value = stateID;

		physx::PxU32 inputID = inStateToInput[ stateID ];
		inputID &= ~NiIosBufferDesc::NEW_PARTICLE_FLAG;
		if (inputID < maxInputID) //this will check also that (inputID != NiIosBufferDesc::NOT_A_PARTICLE)
		{
			if (outputDensityKeys)
			{
				const float4 pos4 = tex1Dfetch(KERNEL_TEX_REF(RemapPositions), inputID);
				const physx::PxVec3 pos = physx::PxVec3(pos4.x, pos4.y, pos4.z);
				const float dist = zNear + (eyePos - pos).dot(eyeDir);
				key = floatFlip( dist );

				//store distance sign in the highest bit of value
				value |= (key & STATE_ID_DIST_SIGN);
			}
			else
			{
				key = getActorIndex(inputID, actorID, numActorClasses, numActorIDs);
			}
		}
		sortKey[ stateID ] = key;
		sortValue[ stateID ] = value;
	}
BOUND_KERNEL_END()


/* Sorted State Space */
BOUND_KERNEL_BEG( REMAP_WARPS_PER_BLOCK, remapKernel,
	const physx::PxU32* inStateToInput, physx::PxU32 maxInputID,
	const NiIofxActorID* actorID, physx::PxU32 numActorClasses, physx::PxU32 numActorIDs,
	const physx::PxU32* inSortedValue, physx::PxU32* outSortKey
)
	const physx::PxU32 maxStateID = _threadCount;
	for (physx::PxU32 stateID = BlockSize*blockIdx.x + threadIdx.x; stateID < maxStateID; stateID += BlockSize*gridDim.x)
	{
		physx::PxU32 actorIndex = ((numActorIDs + 1) << NiIofxActorID::ASSETS_PER_MATERIAL_BITS);

		const physx::PxU32 sortedStateID = (inSortedValue[ stateID ] & STATE_ID_MASK);
		// sortedStateID should be < maxStateID
		physx::PxU32 inputID = tex1Dfetch(KERNEL_TEX_REF(RemapInStateToInput), sortedStateID);
		inputID &= ~NiIosBufferDesc::NEW_PARTICLE_FLAG;
		if (inputID < maxInputID) //this will check also that (inputID != NiIosBufferDesc::NOT_A_PARTICLE)
		{
			actorIndex = getActorIndex(inputID, actorID, numActorClasses, numActorIDs);
		}

		outSortKey[ stateID ] = actorIndex;
	}
BOUND_KERNEL_END()
