#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.


#include "blocksync.cuh"
#include "scan.cuh"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::iofx;
#include "include/sort.h"


typedef unsigned int uint;

//--------------------------------------------------------------------------------------------------

// ================================================================================================
// Flip a float for sorting
//  finds SIGN of fp number.
//  if it's 1 (negative float), it flips all bits
//  if it's 0 (positive float), it flips the sign only
// ================================================================================================
template <bool doFlip>
inline __device__ uint floatFlip(uint f)
{
	if (doFlip)
	{
		uint mask = -int(f >> 31) | 0x80000000;
		return f ^ mask;
	}
	else
		return f;
}

// ================================================================================================
// flip a float back (invert FloatFlip)
//  signed was flipped from above, so:
//  if sign is 1 (negative), it flips the sign bit back
//  if sign is 0 (positive), it flips all bits back
// ================================================================================================
template <bool doFlip>
inline __device__ uint floatUnflip(uint f)
{
	if (doFlip)
	{
		uint mask = ((f >> 31) - 1) | 0x80000000;
		return f ^ mask;
	}
	else
		return f;
}

//--------------------------------------------------------------------------------------------------

template<class T, int maxlevel>
inline __device__ T scanwarp(T val, volatile T* sData)
{
	int idx = 2 * threadIdx.x - (threadIdx.x & (WARP_SIZE - 1));
	sData[idx] = 0;
	idx += WARP_SIZE;
	sData[idx] = val;

	if (0 <= maxlevel) { sData[idx] += sData[idx - 1]; }
	if (1 <= maxlevel) { sData[idx] += sData[idx - 2]; }
	if (2 <= maxlevel) { sData[idx] += sData[idx - 4]; }
	if (3 <= maxlevel) { sData[idx] += sData[idx - 8]; }
	if (4 <= maxlevel) { sData[idx] += sData[idx -16]; }

	return sData[idx] - val;  // convert inclusive -> exclusive
}

inline __device__ uint4 scan4(uint4 idata, volatile uint* sData)
{
	uint idx = threadIdx.x;

	uint4 val4 = idata;
	uint sum[3];
	sum[0] = val4.x;
	sum[1] = val4.y + sum[0];
	sum[2] = val4.z + sum[1];

	uint val = val4.w + sum[2];

	val = scanwarp<uint, 4>(val, sData);
	__syncthreads();

	if ((idx & (WARP_SIZE - 1)) == WARP_SIZE - 1)
	{
		sData[idx >> LOG2_WARP_SIZE] = val + val4.w + sum[2];
	}
	__syncthreads();

	if (idx < WARP_SIZE)
	{
		sData[idx] = scanwarp<uint, 4>(sData[idx], sData);
	}
	__syncthreads();

	val += sData[idx >> LOG2_WARP_SIZE];

	val4.x = val;
	val4.y = val + sum[0];
	val4.z = val + sum[1];
	val4.w = val + sum[2];

	return val4;
}

//--------------------------------------------------------------------------------------------------


template <uint BlockSize>
inline __device__ uint rank(volatile uint* sdata, uint pred)
{
	uint idx = threadIdx.x;

	uint scanCount = scanBlock<uint, AddOP<uint> >(sdata, pred);

	__shared__ uint totalCount;
	if (idx == BlockSize - 1)
	{
		totalCount = scanCount + pred;
	}
	__syncthreads();

	return pred ? scanCount : (totalCount + idx - scanCount);
}


template <uint BlockSize, uint nbits>
inline __device__ void radixSortBlock(volatile uint* sKeys, volatile uint* sValues, uint &key, uint &value, uint startbit)
{
	uint idx = threadIdx.x;

	#pragma unroll
	for(uint shift = startbit; shift < (startbit + nbits); ++shift)
	{
		uint lsb = ((key >> shift) & 0x1) ^ 0x01;

		uint r = rank<BlockSize>(sKeys, lsb);

		sKeys[r] = key;
		sValues[r] = value;

		__syncthreads();

		key = sKeys[idx];
		value = sValues[idx];

		__syncthreads();
	}
}


//--------------------------------------------------------------------------------------------------
#define FIND_RADIX_OFFSETS(sdata) \
		sdata[idx] = (pos < blockEnd) ? ((key >> startbit) & 0xF) : UINT_MAX; \
		if (idx == 0) sdata[BlockSize] = UINT_MAX; \
		if (idx < 16) sRadixStart[idx] = sRadixEnd[idx] = 0; \
		__syncthreads(); \
		if (sdata[idx] != sdata[idx + 1]) { \
			sRadixEnd[sdata[idx]] = idx + 1; \
			if (sdata[idx + 1] < 16) sRadixStart[sdata[idx + 1]] = idx + 1; \
		} \
		__syncthreads(); \

//--------------------------------------------------------------------------------------------------

template <uint BlockSize, uint nbits, bool doFlip>
inline __device__ void radixSortStep1(uint *keys, uint *values, uint *tempKeys, uint *tempValues, uint count, uint startbit, uint* g_temp,
									  volatile uint* sKeys, volatile uint* sValues, const uint blockBeg, const uint blockEnd)
{
	__shared__ uint sRadixStart[16];
	__shared__ uint sRadixEnd[16];
	__shared__ uint sCounters[16];

	const unsigned int idx = threadIdx.x;
	if (idx < 16) sCounters[idx] = 0;

	//sort blocks
	for (uint blockPos = blockBeg; blockPos < blockEnd; blockPos += BlockSize)
	{
		uint pos = blockPos + idx;

		uint key = UINT_MAX;
		uint value = UINT_MAX;
		if (pos < blockEnd)
		{
			key = floatFlip<doFlip>(keys[pos]);
			value = values[pos];
		}

		__syncthreads();
		radixSortBlock<BlockSize, nbits> (sKeys, sValues, key, value, startbit);

		if (pos < blockEnd) 
		{
			tempKeys[pos] = key;
			tempValues[pos] = value;
		}

		FIND_RADIX_OFFSETS(sKeys)

		if (idx < 16) sCounters[idx] += (sRadixEnd[idx] - sRadixStart[idx]);
	}

	__syncthreads();
	if (idx < 16)
	{
		g_temp[gridDim.x*idx + blockIdx.x] = sCounters[idx];
	}
}

inline __device__ void radixSortStep2(uint* g_temp, volatile uint* sdata, uint gridSize)
{
	const uint ScanCount = gridSize * (16 >> 2);

	const unsigned int idx = threadIdx.x;
	uint4 val = (idx < ScanCount) ? ((uint4*)g_temp)[idx] : make_uint4(0, 0, 0, 0);

	val = scan4(val, sdata);

	if (idx < ScanCount) {
		((uint4*)g_temp)[idx] = val;
		__threadfence();
	}
}

template <uint BlockSize, uint nbits, bool doFlip>
inline __device__ void radixSortStep3(uint *keys, uint *values, uint *tempKeys, uint *tempValues, uint count, uint startbit, uint* g_temp,
									  volatile uint* sKeys, volatile uint* sValues, const uint blockBeg, const uint blockEnd)
{
	__shared__ uint sRadixStart[16];
	__shared__ uint sRadixEnd[16];
	__shared__ uint sCounters[16];

	const unsigned int idx = threadIdx.x;
	if (idx < 16)
	{
		sCounters[idx] = g_temp[gridDim.x*idx + blockIdx.x];
	}

	//reorder data in blocks
	for (uint blockPos = blockBeg; blockPos < blockEnd; blockPos += BlockSize)
	{
		uint pos = blockPos + idx;

		uint key = (pos < blockEnd) ? tempKeys[pos] : UINT_MAX;

		FIND_RADIX_OFFSETS(sKeys)

		sKeys[idx] = key;
		sValues[idx] = (pos < blockEnd) ? tempValues[pos] : UINT_MAX;
		__syncthreads();

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 120)
		if (pos < blockEnd)
		{
			uint radix = (sKeys[idx] >> startbit) & 0xF;
			uint globalOffset = sCounters[radix] + idx - sRadixStart[radix];

			keys[globalOffset]   = floatUnflip<doFlip>(sKeys[idx]);
			values[globalOffset] = sValues[idx];
		}
#else
		//manual coalescing
		const uint halfWarpID = idx >> 4;
		if (halfWarpID < 16)
		{
			const uint halfWarpOffset = idx & 0xF;

			const uint startPos = sCounters[halfWarpID];
			const uint endPos   = startPos + (sRadixEnd[halfWarpID] - sRadixStart[halfWarpID]);

			const uint leadingInvalid = startPos & 0xF;

			uint outOffset = (startPos & ~0xF) + halfWarpOffset;
			if (halfWarpOffset >= leadingInvalid && outOffset < endPos)
			{
				uint inOffset0 = sRadixStart[halfWarpID] + (halfWarpOffset - leadingInvalid);
				keys[outOffset]   = floatUnflip<doFlip>(sKeys[inOffset0]);
				values[outOffset] = sValues[inOffset0];
			}

			outOffset += 16;
			uint inOffset  = sRadixStart[halfWarpID] + (16 - leadingInvalid) + halfWarpOffset;

			for (; outOffset < endPos; outOffset += 16, inOffset += 16)
			{
				keys[outOffset]   = floatUnflip<doFlip>(sKeys[inOffset]);
				values[outOffset] = sValues[inOffset];
			}
		}
#endif
		__syncthreads();

		if (idx < 16) sCounters[idx] += (sRadixEnd[idx] - sRadixStart[idx]);
	}
}

#define SORT_STEP_KERNEL_SETUP(count) \
	const unsigned int DataWarpsPerGrid = ((count + WARP_SIZE-1) >> LOG2_WARP_SIZE); \
	const unsigned int DataWarpsPerBlock = (DataWarpsPerGrid + gridDim.x-1) / gridDim.x; \
	const unsigned int DataCountPerBlock = (DataWarpsPerBlock << LOG2_WARP_SIZE); \
	const unsigned int blockBeg = blockIdx.x * DataCountPerBlock; \
	const unsigned int blockEnd = min(blockBeg + DataCountPerBlock, count); \
	__shared__ volatile uint sdata[BlockSize * 2]; \
	volatile uint* sKeys = sdata; \
	volatile uint* sValues = sdata + BlockSize;

template <uint BlockSize, uint nbits, bool doFlip>
__device__ void radixSortStep(uint *keys, uint *values, uint *tempKeys, uint *tempValues, uint count, uint startbit, uint* g_temp)
{
	SORT_STEP_KERNEL_SETUP(count)

	radixSortStep1<BlockSize, nbits, doFlip>(
		keys, values, tempKeys, tempValues, count, startbit, g_temp,
		sKeys, sValues, blockBeg, blockEnd);
	__threadfence();

	BLOCK_SYNC_BEGIN()

	radixSortStep2(g_temp, sdata, gridDim.x);

	BLOCK_SYNC_END()

	radixSortStep3<BlockSize, nbits, doFlip>(
		keys, values, tempKeys, tempValues, count, startbit, g_temp,
		sKeys, sValues, blockBeg, blockEnd);

	{
		__threadfence();
		BLOCK_SYNC_BEGIN()
		BLOCK_SYNC_END()
	}
}

//--------------------------------------------------------------------------------------------------

SYNC_KERNEL_BEG(SORT_WARPS_PER_BLOCK, radixSortKernel, uint numElements,
	uint *keys, uint *values, uint *tempKeys, uint *tempValues, uint* g_temp, uint keyBits, uint startbit0
)
	for (uint startbit = startbit0; startbit < startbit0 + keyBits; startbit += RADIX_SORT_NBITS) {
		radixSortStep<BlockSize, RADIX_SORT_NBITS, false> (keys, values, tempKeys, tempValues, numElements, startbit, g_temp);
	}
SYNC_KERNEL_END()

BOUND_KERNEL_BEG(SORT_WARPS_PER_BLOCK, radixSortStep1Kernel,
	uint *keys, uint *values, uint *tempKeys, uint *tempValues, uint* g_temp, uint startBit
)
	SORT_STEP_KERNEL_SETUP(_threadCount)

	radixSortStep1<BlockSize, RADIX_SORT_NBITS, false>(
		keys, values, tempKeys, tempValues, _threadCount, startBit, g_temp,
		sKeys, sValues, blockBeg, blockEnd);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SORT_WARPS_PER_BLOCK, radixSortStep2Kernel,
	uint* g_temp, uint gridSize
)
	__shared__ volatile uint sdata[BlockSize * 2];

	radixSortStep2(g_temp, sdata, gridSize);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SORT_WARPS_PER_BLOCK, radixSortStep3Kernel,
	uint *keys, uint *values, uint *tempKeys, uint *tempValues, uint* g_temp, uint startBit
)
	SORT_STEP_KERNEL_SETUP(_threadCount)

	radixSortStep3<BlockSize, RADIX_SORT_NBITS, false>(
		keys, values, tempKeys, tempValues, _threadCount, startBit, g_temp,
		sKeys, sValues, blockBeg, blockEnd);

BOUND_KERNEL_END()
