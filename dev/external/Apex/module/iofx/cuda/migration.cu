#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.


#include "common.cuh"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::iofx;
#include "include/migration.h"


__device__
bool contains( const physx::PxBounds3& b, const physx::PxVec3& v )
{
	return !(v.x < b.minimum.x || v.x > b.maximum.x ||
			 v.y < b.minimum.y || v.y > b.maximum.y ||
			 v.z < b.minimum.z || v.z > b.maximum.z);
}

/* Input Space */
BOUND_KERNEL_BEG( VOLUME_MIGRATION_WARPS_PER_BLOCK, volumeMigrationKernel,
	InplaceHandle<VolumeParamsArray> volumeParamsArrayHandle,
	InplaceHandle<ActorIDBitmapArray> actorIDBitmapArrayHandle,
	physx::PxU32 numActorClasses, physx::PxU32 numVolumes,
	NiIofxActorID* actorID, physx::PxU32 maxInputID,
	const float4* positionMass, 
	physx::PxU32* actorStart, physx::PxU32* actorEnd, physx::PxU32* actorVisibleEnd
)
	for (unsigned int input = BlockSize*blockIdx.x + threadIdx.x; input < maxInputID; input += BlockSize*gridDim.x)
	{
		NiIofxActorID id = actorID[ input ];
		const float4 pos4 = positionMass[ input ];
		const physx::PxVec3 pos = physx::PxVec3(pos4.x, pos4.y, pos4.z);

		physx::PxU32 bit = id.getActorClassID();
		if (bit == NiIofxActorID::INV_ACTOR || bit >= numActorClasses)
		{
			id.set( NiIofxActorID::NO_VOLUME, NiIofxActorID::INV_ACTOR );
		}
		else
		{
			physx::PxU32 curPri = 0;
			physx::PxU32 curVID = NiIofxActorID::NO_VOLUME;
			
			const VolumeParamsArray& volumeParamsArray = *volumeParamsArrayHandle.resolve( KERNEL_CONST_MEM(volumeConstMem) );
			//const physx::PxU32 numVolumes = volumeParamsArray.getSize();
			const VolumeParams* volumeParams = volumeParamsArray.getElems( KERNEL_CONST_MEM(volumeConstMem) );

			const ActorIDBitmapArray& actorIDBitmapArray = *actorIDBitmapArrayHandle.resolve( KERNEL_CONST_MEM(volumeConstMem) );
			const physx::PxU32* iofxActorBitmap = actorIDBitmapArray.getElems( KERNEL_CONST_MEM(volumeConstMem) );

			for (physx::PxU32 i = 0 ; i < numVolumes ; i++)
			{
				const physx::PxBounds3& b = volumeParams[i].bounds;
				const physx::PxU32 pri = volumeParams[i].priority;

				// This volume owns this particle if:
				//  1. The volume bounds contain the particle
				//  2. The volume affects the particle's IOFX Asset
				//  3. This volume has the highest priority or was the previous owner
				if ( contains( b, pos ) &&
				     (iofxActorBitmap[ bit >> 5 ] & (1u << (bit & 31))) &&
				     (curVID == NiIofxActorID::NO_VOLUME || pri > curPri || (pri == curPri && id.getVolumeID() == i)) )
				{
					curVID = i;
					curPri = pri;
				}

				bit += numActorClasses;
			}

			id.setVolumeID( curVID );
		}
		actorID[ input ] = id;
	}

	// Clear actorID start/stop table
	const physx::PxU32 numActorIDValues = ((numActorClasses >> NiIofxActorID::ASSETS_PER_MATERIAL_BITS) * numVolumes) + 2;
	for (physx::PxU32 idx = BlockSize*blockIdx.x + threadIdx.x; idx < numActorIDValues; idx += BlockSize*gridDim.x)
	{
		actorStart[ idx ] = 0;
		actorEnd[ idx ] = 0;
		actorVisibleEnd[ idx ] = 0;
	}
		
BOUND_KERNEL_END()
