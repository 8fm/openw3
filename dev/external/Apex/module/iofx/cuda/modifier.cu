#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.


#include "common.cuh"

#include "../public/NxModifierDefs.h"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::iofx;
#include "include/modifier.h"


// Modifiers

#define MODIFIER_DECL __device__
#define CURVE_TYPE Curve
#define EVAL_CURVE(curve, value) curve.evaluate(KERNEL_CONST_MEM(modifierConstMem), value)
#define PARAMS_NAME(name) name ## ParamsGPU

#include "../include/ModifierSrc.h"

#undef MODIFIER_DECL
#undef CURVE_TYPE
#undef EVAL_CURVE
#undef PARAMS_NAME


#define MODIFIER_LIST_BEG(usage) \
template <bool spawn, typename Input, typename PublicState, typename PrivateState> \
__device__ void runModifiers(Usage2Type< usage > , const ModifierList& list, const ModifierCommonParams& commonParams, const Input& input, PublicState& pubState, PrivateState& privState, physx::RandState& randState) \
{ \
	typedef Usage2Type< usage > UsageType; \
	const ModifierListElem* listElems = list.getElems( KERNEL_CONST_MEM(modifierConstMem) ); \
	const physx::PxU32 listCount = list.getSize(); \
	for (unsigned int i = 0; i < listCount; ++i) \
	{ \
		unsigned int type = listElems[i].type; \

#define MODIFIER_LIST_ELEM(name) \
		if (type == physx::apex::ModifierType_##name) \
		{ \
			const name##ParamsGPU & params = *listElems[i].paramsHandle.resolveAndCastTo< name##ParamsGPU >( KERNEL_CONST_MEM(modifierConstMem) ); \
			modifier##name <spawn, UsageType::usage> (params, input, pubState, privState, commonParams, randState); \
		} \
		else \

#define MODIFIER_LIST_END() \
		{ \
		} \
	} \
} \

template <int U>
struct Usage2Type
{
	static const int usage = U;
};

//Sprite modifiers
MODIFIER_LIST_BEG(physx::apex::ModifierUsage_Sprite)
#define _MODIFIER_SPRITE(name) MODIFIER_LIST_ELEM(name)
#include "../include/ModifierList.h"
MODIFIER_LIST_END()

//Mesh modifiers
MODIFIER_LIST_BEG(physx::apex::ModifierUsage_Mesh)
#define _MODIFIER_MESH(name) MODIFIER_LIST_ELEM(name)
#include "../include/ModifierList.h"
MODIFIER_LIST_END()


template <
	int Usage, typename Input, typename PublicState, typename PrivateState,
	typename InputArgs, typename PrivateStateArgs, typename OutputLayout
>
__device__ void modifiersKernel(
	unsigned int outputCount, unsigned int OutputDWords,
	unsigned int inStateOffset, unsigned int outStateOffset,
	InplaceHandle<AssetParamsHandleArray> assetParamsHandleArrayHandle,
	ModifierCommonParams commonParams, unsigned int numActorIDs,
	unsigned int* g_sortedActorIDs, unsigned int* g_sortedStateIDs, unsigned int* g_outStateToInput,
	InputArgs inputArgs, PrivateStateArgs privStateArgs,
	PRNGInfo rand,
	unsigned int* g_outputBuffer, OutputLayout outputLayout
)
{
	unsigned int idx = threadIdx.x;

	const unsigned int BlockSize = blockDim.x;
	const unsigned int Pitch = BlockSize + (NUM_BANKS + OutputDWords-1) / OutputDWords;
	extern __shared__ volatile unsigned int sdata[]; //size = (BlockSize + NUM_BANKS) * outputDWords;

	__shared__ physx::apex::LCG_PRNG randBlock;
	if (idx == 0) {
		randBlock = rand.g_randBlock[blockIdx.x];
	}

	for (unsigned int outputBeg = BlockSize * blockIdx.x; outputBeg < outputCount; outputBeg += BlockSize*gridDim.x)
	{
		physx::apex::LCG_PRNG randVal = (idx == 0 ? randBlock : rand.randThread);
		randVal = randScanBlock(randVal, sdata, sdata + BlockSize*2);

		unsigned int currSeed = randVal(rand.seed);
		if (idx == 0) {
			randBlock *= rand.randGrid;
		}
		__syncthreads();

		const unsigned int outputEnd = min(outputBeg + BlockSize, outputCount);
		const unsigned int outputID = outputBeg + idx;
		if (outputID < outputEnd)
		{
			unsigned int stateID = (g_sortedStateIDs[ outputID ] & STATE_ID_MASK);
			// stateID should be < maxStateID
			unsigned int inputID = tex1Dfetch( KERNEL_TEX_REF(InStateToInput), stateID );
			// inputID should be < maxInputID
			bool isNewParticle = ((inputID & NiIosBufferDesc::NEW_PARTICLE_FLAG) != 0);
			inputID &= ~NiIosBufferDesc::NEW_PARTICLE_FLAG;

			unsigned int actorID = g_sortedActorIDs[ outputID ];
			if (actorID < numActorIDs)
			{
				const AssetParamsHandleArray& assetParamsHandleArray = *assetParamsHandleArrayHandle.resolve( KERNEL_CONST_MEM(modifierConstMem) );
				const physx::PxU32 numberActorClasses = assetParamsHandleArray.getSize();
				const InplaceHandle<AssetParams> assetParamsHandle = assetParamsHandleArray.getElems( KERNEL_CONST_MEM(modifierConstMem) )[ actorID % numberActorClasses ];
				const AssetParams& assetParams = *assetParamsHandle.resolve( KERNEL_CONST_MEM(modifierConstMem) );

				//const ModifierCommonParams& commonParams = *commonParamsHandle.resolve( modifierConstMem );

				//prepare input
				Input		input;
				InputArgs::read(inputArgs, input, inputID, commonParams);

				//prepare state
				PublicState  pubState;
				PrivateState privState;

				//always run spawn modifiers
				PublicState::initDefault(pubState);
				PrivateState::initDefault(privState);

				unsigned int spawnSeed = isNewParticle ? currSeed : tex1Dfetch( KERNEL_TEX_REF(StateSpawnSeed), inStateOffset + stateID );
				RandState spawnRandState( spawnSeed );
				runModifiers<true>(Usage2Type<Usage>(), assetParams.spawnModifierList, commonParams, input, pubState, privState, spawnRandState);

				if (isNewParticle == false)
				{
					//read private state
					PrivateStateArgs::read(privStateArgs, privState, inStateOffset + stateID);
				}

				//run continuous modifiers
				RandState currRandState( currSeed );
				runModifiers<false>(Usage2Type<Usage>(), assetParams.continuousModifierList, commonParams, input, pubState, privState, currRandState);

				//write state
				rand.g_stateSpawnSeed[ outStateOffset + outputID ] = spawnSeed;
				PrivateStateArgs::write(privStateArgs, privState, outStateOffset + outputID);

				//write output to Output
				outputLayout.write(sdata, idx, Pitch, input, pubState, outputID);
			}
			g_outStateToInput[ outputID ] = inputID;
		}
		__syncthreads();

		if (g_outputBuffer != 0)
		{
			const unsigned int OutputBufferDwords = OutputDWords * (outputEnd - outputBeg);
			for (unsigned int pos = threadIdx.x; pos < OutputBufferDwords; pos += BlockSize)
			{
				g_outputBuffer[(outputBeg * OutputDWords) + pos] = sdata[(pos / OutputDWords) + (pos % OutputDWords)*Pitch];
			}
		}
		__syncthreads();
	}
}

// Sprite
namespace physx {
namespace apex {

struct SpriteInputArgs
{
	static __device__ void read(const SpriteInputArgs& args, SpriteInput& input, unsigned int pos, const ModifierCommonParams& commonParams)
	{
		float4 positionMass = tex1Dfetch(KERNEL_TEX_REF(PositionMass), pos);
		float4 velocityLife = tex1Dfetch(KERNEL_TEX_REF(VelocityLife), pos);
		float  density      = commonParams.inputHasDensity ? tex1Dfetch(KERNEL_TEX_REF(Density), pos) : 0;

		input.position.x = positionMass.x;
		input.position.y = positionMass.y;
		input.position.z = positionMass.z;
		input.mass       = positionMass.w;

		input.velocity.x = velocityLife.x;
		input.velocity.y = velocityLife.y;
		input.velocity.z = velocityLife.z;
		input.liferemain = velocityLife.w;

		input.density    = density;

		input.userData   = tex1Dfetch(KERNEL_TEX_REF(UserData), pos);
	}
};

__device__ unsigned int floatFlip(float f)
{
    unsigned int i = __float_as_int(f);
	unsigned int mask = -int(i >> 31) | 0x80000000;
	return i ^ mask;
}


__device__ void SpritePrivateStateArgs::read(const SpritePrivateStateArgs& args, SpritePrivateState& state, unsigned int pos)
{
	IofxSlice slice0 = uint4_to_IofxSlice(tex1Dfetch(KERNEL_TEX_REF(SpritePrivState0), pos));

	// Slice 0 (underused)
	state.rotation = __int_as_float(slice0.x);
}
__device__ void SpritePrivateStateArgs::write(SpritePrivateStateArgs& args, const SpritePrivateState& state, unsigned int pos)
{
	IofxSlice slice0;

	// Slice 0 (underused)
	slice0.x = __float_as_int(state.rotation);

	args.g_state[0][pos] = slice0;
}

// Mesh

struct MeshInputArgs
{
	static __device__ void read(const MeshInputArgs& args, MeshInput& input, unsigned int pos, const ModifierCommonParams& commonParams)
	{
		float4 positionMass         = tex1Dfetch(KERNEL_TEX_REF(PositionMass), pos);
		float4 velocityLife         = tex1Dfetch(KERNEL_TEX_REF(VelocityLife), pos);
		float4 collisionNormalFlags = commonParams.inputHasCollision ? tex1Dfetch(KERNEL_TEX_REF(CollisionNormalFlags), pos) : make_float4(0, 0, 0, 0);
		float  density              = commonParams.inputHasDensity ? tex1Dfetch(KERNEL_TEX_REF(Density), pos) : 0;

		input.position.x = positionMass.x;
		input.position.y = positionMass.y;
		input.position.z = positionMass.z;
		input.mass       = positionMass.w;

		input.velocity.x = velocityLife.x;
		input.velocity.y = velocityLife.y;
		input.velocity.z = velocityLife.z;
		input.liferemain = velocityLife.w;

		input.density    = density;

		input.collisionNormal.x = collisionNormalFlags.x;
		input.collisionNormal.y = collisionNormalFlags.y;
		input.collisionNormal.z = collisionNormalFlags.z;
		input.collisionFlags    = __float_as_int(collisionNormalFlags.w);

		input.userData   = tex1Dfetch(KERNEL_TEX_REF(UserData), pos);
	}
};


__device__ void MeshPrivateStateArgs::read(const MeshPrivateStateArgs& args, MeshPrivateState& state, unsigned int pos)
{
	IofxSlice slice0 = uint4_to_IofxSlice(tex1Dfetch(KERNEL_TEX_REF(MeshPrivState0), pos)),
		slice1 = uint4_to_IofxSlice(tex1Dfetch(KERNEL_TEX_REF(MeshPrivState1), pos)),
		slice2 = uint4_to_IofxSlice(tex1Dfetch(KERNEL_TEX_REF(MeshPrivState2), pos));

	// Slice 0
	state.rotation(0,0) = __int_as_float(slice0.x);
	state.rotation(0,1) = __int_as_float(slice0.y);
	state.rotation(0,2) = __int_as_float(slice0.z);
	state.rotation(1,0) = __int_as_float(slice0.w);

	// Slice 1
	state.rotation(1,1) = __int_as_float(slice1.x);
	state.rotation(1,2) = __int_as_float(slice1.y);
	state.rotation(2,0) = __int_as_float(slice1.z);
	state.rotation(2,1) = __int_as_float(slice1.w);

	// Slice 2 (underused)
	state.rotation(2,2) = __int_as_float(slice2.x);
}
__device__ void MeshPrivateStateArgs::write(MeshPrivateStateArgs& args, const MeshPrivateState& state, unsigned int pos)
{
	IofxSlice slice0, slice1, slice2;
	
	// Slice 0
	slice0.x = __float_as_int(state.rotation(0,0));
	slice0.y = __float_as_int(state.rotation(0,1));
	slice0.z = __float_as_int(state.rotation(0,2));
	slice0.w = __float_as_int(state.rotation(1,0));

	// Slice 1
	slice1.x = __float_as_int(state.rotation(1,1));
	slice1.y = __float_as_int(state.rotation(1,2));
	slice1.z = __float_as_int(state.rotation(2,0));
	slice1.w = __float_as_int(state.rotation(2,1));

	// Slice 2 (underused)
	slice2.x = __float_as_int(state.rotation(2,2));

	args.g_state[0][pos] = slice0;
	args.g_state[1][pos] = slice1;
	args.g_state[2][pos] = slice2;
}


}} // namespace apex

//__launch_bounds__( GET_WARPS_PER_BLOCK(SPRITE_MODIFIER_WARPS_PER_BLOCK) * WARP_SIZE )
BOUND_KERNEL_BEG(SPRITE_MODIFIER_WARPS_PER_BLOCK, spriteModifiersKernel,
	unsigned int inStateOffset, unsigned int outStateOffset,
	InplaceHandle<AssetParamsHandleArray> assetParamsHandleArrayHandle,
	ModifierCommonParams commonParams, unsigned int numActorIDs,
	unsigned int* g_sortedActorIDs, unsigned int* g_sortedStateIDs, unsigned int* g_outStateToInput,
	SpritePrivateStateArgs privStateArgs,
	PRNGInfo rand, unsigned int* g_outputBuffer,
	InplaceHandle<SpriteOutputLayout> outputLayoutHandle
)
	SpriteInputArgs inputArgs;

	const SpriteOutputLayout& outputLayout = *outputLayoutHandle.resolve( KERNEL_CONST_MEM(modifierConstMem) );
	unsigned int OutputDWords = (outputLayout.stride >> 2);

	modifiersKernel<ModifierUsage_Sprite, 
		SpriteInput, SpritePublicState, SpritePrivateState, 
		SpriteInputArgs, SpritePrivateStateArgs, SpriteOutputLayout>
	(
		_threadCount, OutputDWords,
		inStateOffset, outStateOffset,
		assetParamsHandleArrayHandle,
		commonParams, numActorIDs,
		g_sortedActorIDs, g_sortedStateIDs, g_outStateToInput,
		inputArgs, privStateArgs,
		rand,
		g_outputBuffer, outputLayout
	);
BOUND_KERNEL_END()


//__launch_bounds__( GET_WARPS_PER_BLOCK(SPRITE_MODIFIER_WARPS_PER_BLOCK) * WARP_SIZE )
BOUND_KERNEL_BEG(SPRITE_MODIFIER_WARPS_PER_BLOCK, spriteTextureModifiersKernel,
	unsigned int inStateOffset, unsigned int outStateOffset,
	InplaceHandle<AssetParamsHandleArray> assetParamsHandleArrayHandle,
	ModifierCommonParams commonParams, unsigned int numActorIDs,
	unsigned int* g_sortedActorIDs, unsigned int* g_sortedStateIDs, unsigned int* g_outStateToInput,
	SpritePrivateStateArgs privStateArgs,
	PRNGInfo rand, SpriteTextureOutputLayout outputLayout
)
	SpriteInputArgs inputArgs;

	modifiersKernel<ModifierUsage_Sprite, 
		SpriteInput, SpritePublicState, SpritePrivateState, 
		SpriteInputArgs, SpritePrivateStateArgs, SpriteTextureOutputLayout>
	(
		_threadCount, 1,
		inStateOffset, outStateOffset,
		assetParamsHandleArrayHandle,
		commonParams, numActorIDs,
		g_sortedActorIDs, g_sortedStateIDs, g_outStateToInput,
		inputArgs, privStateArgs,
		rand,
		0, outputLayout
	);
BOUND_KERNEL_END()

//__launch_bounds__( GET_WARPS_PER_BLOCK(MESH_MODIFIER_WARPS_PER_BLOCK) * WARP_SIZE )
BOUND_KERNEL_BEG(MESH_MODIFIER_WARPS_PER_BLOCK, meshModifiersKernel,
	unsigned int inStateOffset, unsigned int outStateOffset,
	InplaceHandle<AssetParamsHandleArray> assetParamsHandleArrayHandle,
	ModifierCommonParams commonParams, unsigned int numActorIDs,
	unsigned int* g_sortedActorIDs, unsigned int* g_sortedStateIDs, unsigned int* g_outStateToInput,
	MeshPrivateStateArgs privStateArgs,
	PRNGInfo rand,
	unsigned int* g_outputBuffer,
	InplaceHandle<MeshOutputLayout> outputLayoutHandle
)
	MeshInputArgs inputArgs;

	const MeshOutputLayout& outputLayout = *outputLayoutHandle.resolve( KERNEL_CONST_MEM(modifierConstMem) );
	unsigned int OutputDWords = (outputLayout.stride >> 2);

	modifiersKernel<ModifierUsage_Mesh,
		MeshInput, MeshPublicState, MeshPrivateState,
		MeshInputArgs, MeshPrivateStateArgs, MeshOutputLayout>
	(
		_threadCount, OutputDWords,
		inStateOffset, outStateOffset,
		assetParamsHandleArrayHandle,
		commonParams, numActorIDs,
		g_sortedActorIDs, g_sortedStateIDs, g_outStateToInput,
		inputArgs, privStateArgs,
		rand,
		g_outputBuffer, outputLayout
	);
BOUND_KERNEL_END()
