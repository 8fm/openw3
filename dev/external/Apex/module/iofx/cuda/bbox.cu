#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.

#define FIX_FOR_KEPLER 1

#include "blocksync.cuh"
#include "reduce.cuh"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::iofx;
#include "include/bbox.h"


#define DEBUG_BBOX2 0

#if DEBUG_BBOX2
	typedef AddOPf OP;
	#define OUTPUT_DEBUG_INFO(pos, a1, a2, a3, a4) ((float4*)g_positionMass)[pos] = make_float4(a1, a2, a3, a4);
#else
	typedef MinOPf OP;
	#define OUTPUT_DEBUG_INFO(pos, a1, a2, a3, a4)
#endif

#define REDUCE_STEP(sdata, pitch, n) \
	sdata[pitch*0 + idx] = OP::apply(sdata[pitch*0 + idx], sdata[pitch*0 + idx + n]); \
	sdata[pitch*1 + idx] = OP::apply(sdata[pitch*1 + idx], sdata[pitch*1 + idx + n]); \
	sdata[pitch*2 + idx] = OP::apply(sdata[pitch*2 + idx], sdata[pitch*2 + idx + n]); \
	sdata[pitch*3 + idx] = OP::apply(sdata[pitch*3 + idx], sdata[pitch*3 + idx + n]); \
	sdata[pitch*4 + idx] = OP::apply(sdata[pitch*4 + idx], sdata[pitch*4 + idx + n]); \
	sdata[pitch*5 + idx] = OP::apply(sdata[pitch*5 + idx], sdata[pitch*5 + idx + n]);

#define REDUCE_BLOCK_WHOLE_WARP() \
	if (idxInWarp < 16) { \
		REDUCE_STEP(sBlockBounds, BlockPitch, 16) \
		REDUCE_STEP(sBlockBounds, BlockPitch, 8) \
		REDUCE_STEP(sBlockBounds, BlockPitch, 4) \
		REDUCE_STEP(sBlockBounds, BlockPitch, 2) \
		REDUCE_STEP(sBlockBounds, BlockPitch, 1) \
	}

#define REDUCE_BLOCK_STEP(n) \
	if (idxInWarp + n < WARP_SIZE && sBlockActorID[idx + n] == sBlockActorID[idx]) \
	{ \
		REDUCE_STEP(sBlockBounds, BlockPitch, n) \
	}

#define REDUCE_WARP_STEP(n) \
	if (idx + n < WARP_SIZE && sWarpLastActorID[idx + n] == sWarpLastActorID[idx]) \
	{ \
		sWarpSegmentSize[idx] += sWarpSegmentSize[idx + n]; \
		REDUCE_STEP(sWarpLastBounds, WarpPitch, n) \
	}

#define INPUT_BOUNDS(dst, pitch, idx, vmin, vmax) \
	dst[pitch*0 + idx] = vmin.x; \
	dst[pitch*1 + idx] = vmin.y; \
	dst[pitch*2 + idx] = vmin.z; \
	dst[pitch*3 + idx] = -vmax.x; \
	dst[pitch*4 + idx] = -vmax.y; \
	dst[pitch*5 + idx] = -vmax.z;

#define INPUT_OP_BOUNDS(dst, pitch, idx, vmin, vmax) \
	dst[pitch*0 + idx] = OP::apply(dst[pitch*0 + idx], vmin.x); \
	dst[pitch*1 + idx] = OP::apply(dst[pitch*1 + idx], vmin.y); \
	dst[pitch*2 + idx] = OP::apply(dst[pitch*2 + idx], vmin.z); \
	dst[pitch*3 + idx] = OP::apply(dst[pitch*3 + idx], -vmax.x); \
	dst[pitch*4 + idx] = OP::apply(dst[pitch*4 + idx], -vmax.y); \
	dst[pitch*5 + idx] = OP::apply(dst[pitch*5 + idx], -vmax.z);

#define OUTPUT_BOUNDS(dst, pos, src, pitch, idx) \
	dst##MinBounds[pos] = make_float4( \
		src[pitch*0 + idx], \
		src[pitch*1 + idx], \
		src[pitch*2 + idx], 0); \
	dst##MaxBounds[pos] = make_float4( \
		-src[pitch*3 + idx], \
		-src[pitch*4 + idx], \
		-src[pitch*5 + idx], 0);

#define RESET_LAST_BOUNDS() \
	if (idxInWarp < 6) { \
		sWarpLastBounds[WarpPitch*idxInWarp + warpIdx] = OP::identity(); \
	}

#define UPDATE_LAST_BOUNDS() \
	if (idxInWarp < 6) { \
		sWarpLastBounds[WarpPitch*idxInWarp + warpIdx] = OP::apply( \
			sWarpLastBounds[WarpPitch*idxInWarp + warpIdx], sBlockBounds[BlockPitch*idxInWarp + warpFirstIdx]); \
	}

#define WRITE_LAST_BOUNDS() \
	if (firstActorID != UINT_MAX) { \
		if (idxInWarp == 0) { \
			OUTPUT_BOUNDS(g_out, lastActorID, sWarpLastBounds, WarpPitch, warpIdx) \
			OUTPUT_DEBUG_INFO(lastActorID, 0, blockIdx.x, warpIdx, pos) \
		} \
	} else { \
		firstActorID = lastActorID; \
		if (idxInWarp < 6) { \
			sWarpFirstBounds[WarpPitch*idxInWarp + warpIdx] = sWarpLastBounds[WarpPitch*idxInWarp + warpIdx]; \
		} \
	}


template <unsigned int WarpsPerBlock, unsigned int BlockSize, unsigned int BlockPitch, unsigned int WarpPitch>
inline __device__ void bbox1(
	unsigned int count,
	unsigned int* g_actorID,
	unsigned int* stateToInput,
	const float4* g_positionMass,
	float4* g_outMinBounds, float4* g_outMaxBounds,
	unsigned int* g_tmpLastActorID, unsigned int* g_tmpFirstActorID,
	float4* g_tmpLastMinBounds, float4* g_tmpFirstMinBounds,
	float4* g_tmpLastMaxBounds, float4* g_tmpFirstMaxBounds,
	volatile unsigned int* sBlockActorID, volatile float* sBlockBounds,
	volatile unsigned int* sWarpLastActorID, volatile float* sWarpLastBounds,
	volatile unsigned int* sWarpFirstActorID, volatile float* sWarpFirstBounds)
{
	const unsigned int DataWarpsPerGrid = ((count + WARP_SIZE-1) >> LOG2_WARP_SIZE);
	const unsigned int DataWarpsPerBlock = (DataWarpsPerGrid + gridDim.x-1) / gridDim.x;
	const unsigned int DataCountPerBlock = (DataWarpsPerBlock << LOG2_WARP_SIZE);

	const unsigned int WarpLimit = min(DataWarpsPerBlock, WarpsPerBlock);
	const unsigned int WarpBorder = DataWarpsPerBlock % WarpsPerBlock;
	const unsigned int WarpFactor = DataWarpsPerBlock / WarpsPerBlock;

	const unsigned int idx = threadIdx.x;
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	const unsigned int blockBeg = blockIdx.x * DataCountPerBlock;
	const unsigned int blockEnd = min(blockBeg + DataCountPerBlock, count);

	const unsigned int WarpSelect = (warpIdx < WarpBorder) ? 1 : 0;
	const unsigned int WarpCount = WarpFactor + WarpSelect;
	const unsigned int WarpOffset = warpIdx * WarpCount + WarpBorder * (1 - WarpSelect);

	const unsigned int warpBeg = blockBeg + (WarpOffset << LOG2_WARP_SIZE);
	const unsigned int warpEnd = min(warpBeg + (WarpCount << LOG2_WARP_SIZE), blockEnd);

	const unsigned int idxInWarp = idx & (WARP_SIZE-1);


	unsigned int lastActorID = UINT_MAX;
	unsigned int firstActorID = UINT_MAX;
	if (warpBeg < warpEnd)
	{
		unsigned int lastWholeWarp = 0;

		const unsigned int warpFirstIdx = idx & ~(WARP_SIZE-1);

		unsigned int pos;
		for (pos = warpBeg + idxInWarp; pos < (warpEnd & ~(WARP_SIZE-1)); pos += WARP_SIZE)
		{
			//read data
			unsigned int actorID = (g_actorID[pos] >> NiIofxActorID::ASSETS_PER_MATERIAL_BITS);
			sBlockActorID[idx] = actorID;

			const unsigned int warpFirstActorID = sBlockActorID[warpFirstIdx];
			const unsigned int warpLastActorID = sBlockActorID[warpFirstIdx + (WARP_SIZE - 1)];

#if DEBUG_BBOX2
			const float4 point = make_float4(1, 2, 3, 0);
#else
			unsigned int input = stateToInput[ pos ];
			const float4 point = tex1Dfetch(KERNEL_TEX_REF(BBoxPositions), input);
#endif
			if (lastWholeWarp)
			{
				if (warpLastActorID == lastActorID) {
					//current warp is also whole - just accum. whole warp
					INPUT_OP_BOUNDS(sBlockBounds, BlockPitch, idx, point, point)
					continue;
				}

				//we have not whole warp - reduce last whole warp
				REDUCE_BLOCK_WHOLE_WARP()
				//and update the last data
				UPDATE_LAST_BOUNDS()
				lastWholeWarp = 0;
			}

			INPUT_BOUNDS(sBlockBounds, BlockPitch, idx, point, point)

			if (warpFirstActorID == warpLastActorID) {
				//we have a whole warp
				if (warpFirstActorID != lastActorID) {
					if (lastActorID != UINT_MAX) {
						WRITE_LAST_BOUNDS()
					}
					//reset last bounds
					RESET_LAST_BOUNDS()
					lastActorID = actorID; //actorID is the same for the whole warp!
				}
				lastWholeWarp = 1;
				continue;
			}

			//reduce
			REDUCE_BLOCK_STEP(1)
			REDUCE_BLOCK_STEP(2)
			REDUCE_BLOCK_STEP(4)
			REDUCE_BLOCK_STEP(8)
			REDUCE_BLOCK_STEP(16)

			if (lastActorID == UINT_MAX) {
				RESET_LAST_BOUNDS()
				lastActorID = warpFirstActorID;
			}
			//update the last data
			if (warpFirstActorID == lastActorID) {
				UPDATE_LAST_BOUNDS()
			}
			//and write it
			WRITE_LAST_BOUNDS()

			unsigned int prevActorID = (idxInWarp > 0) ? sBlockActorID[idx - 1] : lastActorID;
			if (prevActorID != actorID) {
				if (actorID == warpLastActorID) {
					sWarpLastActorID[warpIdx] = idx;
#if FIX_FOR_KEPLER
					sWarpLastBounds[WarpPitch*0 + warpIdx] = sBlockBounds[BlockPitch*0 + idx];
					sWarpLastBounds[WarpPitch*1 + warpIdx] = sBlockBounds[BlockPitch*1 + idx];
					sWarpLastBounds[WarpPitch*2 + warpIdx] = sBlockBounds[BlockPitch*2 + idx];
					sWarpLastBounds[WarpPitch*3 + warpIdx] = sBlockBounds[BlockPitch*3 + idx];
					sWarpLastBounds[WarpPitch*4 + warpIdx] = sBlockBounds[BlockPitch*4 + idx];
					sWarpLastBounds[WarpPitch*5 + warpIdx] = sBlockBounds[BlockPitch*5 + idx];
#endif
				} else {
					OUTPUT_BOUNDS(g_out, actorID, sBlockBounds, BlockPitch, idx)
					OUTPUT_DEBUG_INFO(actorID, 1, blockIdx.x, warpIdx, pos)
				}
			}
#if !FIX_FOR_KEPLER
			//set the last data
			if (idxInWarp < 6) {
				sWarpLastBounds[WarpPitch*idxInWarp + warpIdx] = sBlockBounds[BlockPitch*idxInWarp + sWarpLastActorID[warpIdx]];
			}
#endif
			lastActorID = warpLastActorID;
		}

		if (lastWholeWarp)
		{
			//reduce last whole warp
			REDUCE_BLOCK_WHOLE_WARP()
			//and update the last data
			UPDATE_LAST_BOUNDS()
			//lastWholeWarp = 0;
		}

		//handle the last non-whole warp
		unsigned int lastWarpSize = warpEnd & (WARP_SIZE-1);
		if (lastWarpSize > 0)
		{
			unsigned int actorID = (pos < warpEnd) ? (g_actorID[pos] >> NiIofxActorID::ASSETS_PER_MATERIAL_BITS) : UINT_MAX;
			sBlockActorID[idx] = actorID;

			const unsigned int warpFirstActorID = sBlockActorID[warpFirstIdx];
			const unsigned int warpLastActorID = sBlockActorID[warpFirstIdx + (lastWarpSize - 1)];

			if (pos < warpEnd) {
				//read data
#if DEBUG_BBOX2
				const float4 point = make_float4(1, 2, 3, 0);
#else
				unsigned int input = stateToInput[ pos ];
				const float4 point = tex1Dfetch(KERNEL_TEX_REF(BBoxPositions), input);
#endif
				INPUT_BOUNDS(sBlockBounds, BlockPitch, idx, point, point)
			}

			//reduce
			REDUCE_BLOCK_STEP(1)
			REDUCE_BLOCK_STEP(2)
			REDUCE_BLOCK_STEP(4)
			REDUCE_BLOCK_STEP(8)
			REDUCE_BLOCK_STEP(16)

			if (lastActorID == UINT_MAX) {
				RESET_LAST_BOUNDS()
				lastActorID = warpFirstActorID;
			}
			if (warpFirstActorID == lastActorID) {
				//update the last data
				UPDATE_LAST_BOUNDS()
			}
			if (warpLastActorID != lastActorID) {
				WRITE_LAST_BOUNDS()

				unsigned int prevActorID = (idxInWarp > 0) ? sBlockActorID[idx - 1] : lastActorID;
				if (actorID != UINT_MAX && prevActorID != actorID) {
					if (actorID == warpLastActorID) {
						sWarpLastActorID[warpIdx] = idx;
					} else {
						OUTPUT_BOUNDS(g_out, actorID, sBlockBounds, BlockPitch, idx)
						OUTPUT_DEBUG_INFO(actorID, 1, blockIdx.x, warpIdx, pos)
					}
				}
				//set the last data
				if (idxInWarp < 6) {
					sWarpLastBounds[WarpPitch*idxInWarp + warpIdx] = sBlockBounds[BlockPitch*idxInWarp + sWarpLastActorID[warpIdx]];
				}
				lastActorID = warpLastActorID;
			}
		}
	}
	if (idxInWarp == 0) {
		sWarpLastActorID[warpIdx] = lastActorID;
		sWarpFirstActorID[warpIdx] = firstActorID;
	}
	//set the last guard values
	if (idx >= WarpLimit && idx <= WARP_SIZE)
	{
		sWarpLastActorID[idx] = UINT_MAX;
		sWarpFirstActorID[idx] = UINT_MAX;
	}
	__syncthreads();

	volatile unsigned int* sWarpSegmentSize = sBlockActorID;

	//check for empty block
	if (sWarpLastActorID[0] != UINT_MAX)
	{
		//one warp
		if (idx < WARP_SIZE)
		{
			sWarpSegmentSize[idx] = 1;

			REDUCE_WARP_STEP(1)
			REDUCE_WARP_STEP(2)
			REDUCE_WARP_STEP(4)
			REDUCE_WARP_STEP(8)
			REDUCE_WARP_STEP(16)

			float4* outMinBounds = 0;
			float4* outMaxBounds = 0;

			//combine with the first bounds of next segment
			unsigned int actorID = sWarpLastActorID[idx];
			unsigned int prevActorID = (idx > 0) ? sWarpLastActorID[idx - 1] : UINT_MAX;
			if (actorID != UINT_MAX && actorID != prevActorID)
			{
				unsigned int nextSegmentIdx = idx + sWarpSegmentSize[idx];
				if (sWarpFirstActorID[nextSegmentIdx] == actorID) {
					#pragma unroll
					for (int i = 0; i < 6; ++i) {
						sWarpLastBounds[WarpPitch*i + idx] = OP::apply(sWarpLastBounds[WarpPitch*i + idx],
							sWarpFirstBounds[WarpPitch*i + nextSegmentIdx]);
					}
					sWarpFirstActorID[nextSegmentIdx] = UINT_MAX;
				}

				if (sWarpLastActorID[nextSegmentIdx] == UINT_MAX) {
					//last segment
					outMinBounds = g_tmpLastMinBounds + blockIdx.x;
					outMaxBounds = g_tmpLastMaxBounds + blockIdx.x;
					g_tmpLastActorID[blockIdx.x] = actorID;
				}
				else
				{
					outMinBounds = g_outMinBounds + actorID;
					outMaxBounds = g_outMaxBounds + actorID;
					OUTPUT_DEBUG_INFO(actorID, 3, blockIdx.x, idx, -1)
				}
			}

			if (sWarpFirstActorID[0] == UINT_MAX && sWarpLastActorID[sWarpSegmentSize[0]] != UINT_MAX)
			{
				if (idx < 6) {
					sWarpFirstBounds[WarpPitch*idx + 0] = sWarpLastBounds[WarpPitch*idx + 0];
				}
				if (idx == 0) {
					sWarpFirstActorID[0] = sWarpLastActorID[0];
					//exclude output of the last bounds for 0 thread
					outMinBounds = outMaxBounds = 0;
				}
			}
			//output last bounds
			if (outMinBounds != 0)
			{
				OUTPUT_BOUNDS(out, 0, sWarpLastBounds, WarpPitch, idx)
			}

			//output first bounds
			actorID = sWarpFirstActorID[idx];
			if (idx == 0) {
				g_tmpFirstActorID[blockIdx.x] = actorID;
			}
			if (actorID != UINT_MAX) {
				if (idx == 0) {
					outMinBounds = g_tmpFirstMinBounds + blockIdx.x;
					outMaxBounds = g_tmpFirstMaxBounds + blockIdx.x;
				} else {
					outMinBounds = g_outMinBounds + actorID;
					outMaxBounds = g_outMaxBounds + actorID;
					OUTPUT_DEBUG_INFO(actorID, 4, blockIdx.x, idx, -1)
				}
				OUTPUT_BOUNDS(out, 0, sWarpFirstBounds, WarpPitch, idx)
			}
		}
	}
	else
	{
		if (idx == 0) {
			g_tmpLastActorID[blockIdx.x] = UINT_MAX;
			g_tmpFirstActorID[blockIdx.x] = UINT_MAX;
		}
	}
}

template <unsigned int WarpsPerBlock, unsigned int BlockSize, unsigned int BlockPitch, unsigned int WarpPitch>
inline __device__ void bbox2(
	unsigned int* g_actorID,
	unsigned int* stateToInput,
	const float4* g_positionMass,
	float4* g_outMinBounds, float4* g_outMaxBounds,
	unsigned int* g_tmpLastActorID, unsigned int* g_tmpFirstActorID,
	float4* g_tmpLastMinBounds, float4* g_tmpFirstMinBounds,
	float4* g_tmpLastMaxBounds, float4* g_tmpFirstMaxBounds,
	volatile unsigned int* sBlockActorID, volatile float* sBlockBounds,
	volatile unsigned int* sWarpLastActorID, volatile float* sWarpLastBounds,
	volatile unsigned int* sWarpFirstActorID, volatile float* sWarpFirstBounds,
	unsigned int gridSize)
{
	const unsigned int idx = threadIdx.x;

	if (idx <= WARP_SIZE)
	{
		sWarpLastActorID[idx] = (idx < gridSize) ? g_tmpLastActorID[idx] : UINT_MAX;
		sWarpFirstActorID[idx] = (idx < gridSize) ? g_tmpFirstActorID[idx] : UINT_MAX;
	}
	__syncthreads();

	volatile unsigned int* sWarpSegmentSize = sBlockActorID;

	//one warp
	if (idx < WARP_SIZE)
	{
		if (idx < gridSize)
		{
			float4 minBound = g_tmpLastMinBounds[idx];
			float4 maxBound = g_tmpLastMaxBounds[idx];
			INPUT_BOUNDS(sWarpLastBounds, WarpPitch, idx, minBound, maxBound)

			minBound = g_tmpFirstMinBounds[idx];
			maxBound = g_tmpFirstMaxBounds[idx];
			INPUT_BOUNDS(sWarpFirstBounds, WarpPitch, idx, minBound, maxBound)
		}

		sWarpSegmentSize[idx] = 1;

		REDUCE_WARP_STEP(1)
		REDUCE_WARP_STEP(2)
		REDUCE_WARP_STEP(4)
		REDUCE_WARP_STEP(8)
		REDUCE_WARP_STEP(16)

		unsigned int actorID = sWarpLastActorID[idx];
		unsigned int prevActorID = (idx > 0) ? sWarpLastActorID[idx - 1] : UINT_MAX;
		if (actorID != UINT_MAX && actorID != prevActorID)
		{
			float bounds[6];
			#pragma unroll
			for (int i = 0; i < 6; ++i) {
				bounds[i] = sWarpLastBounds[WarpPitch*i + idx];
			}

			unsigned int nextSegmentIdx = idx + sWarpSegmentSize[idx];
			if (sWarpFirstActorID[nextSegmentIdx] == actorID) {
				//combine with the first bounds of next segment
				#pragma unroll
				for (int i = 0; i < 6; ++i) {
					bounds[i] = OP::apply(bounds[i], sWarpFirstBounds[WarpPitch*i + nextSegmentIdx]);
				}
				sWarpFirstActorID[nextSegmentIdx] = UINT_MAX;
			}
			OUTPUT_BOUNDS(g_out, actorID, bounds, 1, 0)
			OUTPUT_DEBUG_INFO(actorID, 5, idx, sWarpLastBounds[idx], sWarpFirstBounds[nextSegmentIdx])
		}

		actorID = sWarpFirstActorID[idx];
		if (actorID != UINT_MAX) {
			OUTPUT_BOUNDS(g_out, actorID, sWarpFirstBounds, WarpPitch, idx)
			OUTPUT_DEBUG_INFO(actorID, 6, idx, -1, -1)
		}
	}
}

#define BBOX_KERNEL_SETUP() \
	const unsigned int BlockPitch = BlockSize + 1; \
	__shared__ volatile unsigned int    sBlockActorID[BlockSize]; \
	__shared__ volatile float           sBlockBounds[BlockPitch * 6]; \
	const unsigned int WarpPitch = WARP_SIZE + 1; /*MaxGridSize is 32*/ \
	__shared__ volatile unsigned int    sWarpLastActorID[WARP_SIZE + 1]; \
	__shared__ volatile float           sWarpLastBounds[WarpPitch * 6]; \
	__shared__ volatile unsigned int    sWarpFirstActorID[WARP_SIZE + 1]; \
	__shared__ volatile float           sWarpFirstBounds[WarpPitch * 6]; \
	unsigned int* g_tmpLastActorID = g_tmpActorID; \
	unsigned int* g_tmpFirstActorID = g_tmpActorID + WARP_SIZE; \
	float4* g_tmpLastMinBounds = g_tmpMinBounds; \
	float4* g_tmpFirstMinBounds = g_tmpMinBounds + WARP_SIZE; \
	float4* g_tmpLastMaxBounds = g_tmpMaxBounds; \
	float4* g_tmpFirstMaxBounds = g_tmpMaxBounds + WARP_SIZE;


//g_tmpActorID, g_tmpMinBounds, g_tmpMaxBounds should have size = WARP_SIZE*2!!!
SYNC_KERNEL_BEG(BBOX_WARPS_PER_BLOCK, bboxKernel,
	unsigned int count,
	unsigned int* g_actorID,
	unsigned int* stateToInput,
	const float4* g_positionMass,
	float4* g_outMinBounds, float4* g_outMaxBounds,
	unsigned int* g_tmpActorID,
	float4* g_tmpMinBounds, float4* g_tmpMaxBounds
)
	BBOX_KERNEL_SETUP()

	bbox1<WarpsPerBlock, BlockSize, BlockPitch, WarpPitch>(
		count, g_actorID, stateToInput, g_positionMass,
		g_outMinBounds, g_outMaxBounds,
		g_tmpLastActorID, g_tmpFirstActorID,
		g_tmpLastMinBounds, g_tmpFirstMinBounds,
		g_tmpLastMaxBounds, g_tmpFirstMaxBounds,
		sBlockActorID, sBlockBounds, sWarpLastActorID, sWarpLastBounds, sWarpFirstActorID, sWarpFirstBounds);

	__threadfence();
	BLOCK_SYNC_BEGIN()

	bbox2<WarpsPerBlock, BlockSize, BlockPitch, WarpPitch>(
		g_actorID, stateToInput, g_positionMass,
		g_outMinBounds, g_outMaxBounds,
		g_tmpLastActorID, g_tmpFirstActorID,
		g_tmpLastMinBounds, g_tmpFirstMinBounds,
		g_tmpLastMaxBounds, g_tmpFirstMaxBounds,
		sBlockActorID, sBlockBounds, sWarpLastActorID, sWarpLastBounds, sWarpFirstActorID, sWarpFirstBounds,
		gridDim.x);

	BLOCK_SYNC_END()

SYNC_KERNEL_END()

BOUND_KERNEL_BEG(BBOX_WARPS_PER_BLOCK, bbox1Kernel,
	unsigned int* g_actorID,
	unsigned int* stateToInput,
	const float4* g_positionMass,
	float4* g_outMinBounds, float4* g_outMaxBounds,
	unsigned int* g_tmpActorID,
	float4* g_tmpMinBounds, float4* g_tmpMaxBounds
)
	BBOX_KERNEL_SETUP()

	bbox1<WarpsPerBlock, BlockSize, BlockPitch, WarpPitch>(_threadCount,
		g_actorID, stateToInput, g_positionMass,
		g_outMinBounds, g_outMaxBounds,
		g_tmpLastActorID, g_tmpFirstActorID,
		g_tmpLastMinBounds, g_tmpFirstMinBounds,
		g_tmpLastMaxBounds, g_tmpFirstMaxBounds,
		sBlockActorID, sBlockBounds, sWarpLastActorID, sWarpLastBounds, sWarpFirstActorID, sWarpFirstBounds);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(BBOX_WARPS_PER_BLOCK, bbox2Kernel,
	unsigned int* g_actorID,
	unsigned int* stateToInput,
	const float4* g_positionMass,
	float4* g_outMinBounds, float4* g_outMaxBounds,
	unsigned int* g_tmpActorID,
	float4* g_tmpMinBounds, float4* g_tmpMaxBounds,
	unsigned int gridSize
)
	BBOX_KERNEL_SETUP()

	bbox2<WarpsPerBlock, BlockSize, BlockPitch, WarpPitch>(
		g_actorID, stateToInput, g_positionMass,
		g_outMinBounds, g_outMaxBounds,
		g_tmpLastActorID, g_tmpFirstActorID,
		g_tmpLastMinBounds, g_tmpFirstMinBounds,
		g_tmpLastMaxBounds, g_tmpFirstMaxBounds,
		sBlockActorID, sBlockBounds, sWarpLastActorID, sWarpLastBounds, sWarpFirstActorID, sWarpFirstBounds,
		gridSize);

BOUND_KERNEL_END()
