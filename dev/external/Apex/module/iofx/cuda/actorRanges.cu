#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.


#include "common.cuh"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::iofx;
#include "include/actorRanges.h"
#include "NiIofxManager.h"


BOUND_KERNEL_BEG( ACTOR_RANGE_WARPS_PER_BLOCK, actorRangeKernel,
	const physx::PxU32* sortedActorID, physx::PxU32 numActorIDs,
	physx::PxU32* actorStart, physx::PxU32* actorEnd, physx::PxU32* actorVisibleEnd,
	const physx::PxU32* sortedStateID
)
	__shared__ physx::PxU32 sdata[BlockSize + 1];
	__shared__ physx::PxU32 sdataVisible[BlockSize + 1];

	const physx::PxU32 idx = threadIdx.x;

	const physx::PxU32 outputCount = _threadCount;
	for (unsigned int outputBeg = BlockSize * blockIdx.x; outputBeg < outputCount; outputBeg += BlockSize * gridDim.x)
	{
		const unsigned int outputEnd = min(outputBeg + BlockSize, outputCount);
		const unsigned int output = outputBeg + idx;

		sdata[idx] = (output < outputEnd) ? (sortedActorID[output] >> NiIofxActorID::ASSETS_PER_MATERIAL_BITS) : UINT_MAX;
		sdataVisible[idx] = (output < outputEnd) ? (sortedStateID[output] >> 31) : UINT_MAX;
		if (idx == 0) {
			sdata[BlockSize] = (outputEnd < outputCount) ? (sortedActorID[outputEnd] >> NiIofxActorID::ASSETS_PER_MATERIAL_BITS) : UINT_MAX;
			sdataVisible[BlockSize] = (outputEnd < outputCount) ? (sortedStateID[outputEnd] >> 31) : UINT_MAX;
		}
		__syncthreads();

		if (output < outputEnd)
		{
			const physx::PxU32 currActorIndex = sdata[idx];
			const physx::PxU32 nextActorIndex = sdata[idx + 1];
			if (nextActorIndex != currActorIndex)
			{
				if (nextActorIndex != UINT_MAX)
				{
					actorStart[nextActorIndex] = output + 1;
					if (sdataVisible[idx + 1] != 0)
					{
						actorVisibleEnd[nextActorIndex] = output + 1;
					}
				}
				if (currActorIndex != UINT_MAX)
				{
					actorEnd[currActorIndex] = output + 1;
					if (sdataVisible[idx] == 0)
					{
						actorVisibleEnd[currActorIndex] = output + 1;
					}
				}
			}
			else if (sdataVisible[idx] != sdataVisible[idx + 1])
			{
				if (currActorIndex != UINT_MAX)
				{
					actorVisibleEnd[currActorIndex] = output + 1;
				}
			}
		}
	}
BOUND_KERNEL_END()
