#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.


#include "blocksync.cuh"

#include "include/common.h"
#include "include/histogram.h"

const unsigned int TAG_SHIFT = (32 - LOG2_WARP_SIZE);
const unsigned int TAG_MASK = (1U << TAG_SHIFT) - 1;

typedef volatile unsigned int histogram_t;

inline __device__ void addToBin(histogram_t *s_WarpHist, unsigned int data, unsigned int threadTag)
{
	unsigned int count;
	do {
		count = s_WarpHist[data] & TAG_MASK;
		count = threadTag | (count + 1);
		s_WarpHist[data] = count;
	} while (s_WarpHist[data] != count);
}


template <int WarpsPerBlock, int BlockSize, int BinCount>
inline __device__ void histogram1(unsigned int count,
	const float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms,
	histogram_t* s_Hist
)
{
	const unsigned int idx = threadIdx.x;

	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	histogram_t* s_WarpHist = s_Hist + warpIdx * BinCount;

	//Clear shared memory storage for current threadblock before processing
	#pragma unroll
	for(unsigned int i = 0; i < (BinCount >> LOG2_WARP_SIZE); i++) {
	   s_Hist[idx + i * BlockSize] = 0;
	}

	__syncthreads();

	const unsigned int tag = (idx & (WARP_SIZE-1)) << TAG_SHIFT;

	for(unsigned int pos = (BlockSize*blockIdx.x + idx); pos < count; pos += BlockSize*gridDim.x)
	{
		float data = g_data[pos];
		if (data >= dataMin && data < dataMax)
		{
			unsigned int bin = (data - dataMin)*BinCount/(dataMax - dataMin);
			addToBin(s_WarpHist, bin, tag);
		}
	}

	//Merge per-warp histograms into per-block and write to global memory
	__syncthreads();
	if (idx < BinCount)
	{
		unsigned int sum = 0;

		for(unsigned int i = 0; i < WarpsPerBlock; i++)
			sum += s_Hist[idx + i * BinCount] & TAG_MASK;

		g_tmpHistograms[blockIdx.x * BinCount + idx] = sum;
		__threadfence();
	}
}

template <int WarpsPerBlock, int BlockSize, int BinCount>
inline __device__ void histogram2(
	const float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms,
	histogram_t* s_Hist, unsigned int gridSize
)
{
	const unsigned int idx = threadIdx.x;

	if (idx < BinCount)
	{
		s_Hist[idx] = 0;
		for (unsigned int i = 0; i < gridSize; ++i)
		{
			s_Hist[idx] += g_tmpHistograms[i*BinCount + idx];
		}
	}
	__syncthreads();

	//build CDF using prefix sum
	int pout = 0;
	int pin = 1;

	#pragma unroll
	for (int offset = 1; offset < BinCount; offset *= 2)
	{
		pout = 1 - pout;
		pin  = 1 - pout;

		if (idx < BinCount)
		{
			s_Hist[pout*BinCount + idx] = s_Hist[pin*BinCount + idx];
			if (idx >= offset)
				s_Hist[pout*BinCount + idx] += s_Hist[pin*BinCount + idx - offset];
#ifdef APEX_TEST
			g_tmpHistograms[pout*BinCount + idx] = s_Hist[pout*BinCount + idx];
#endif
		}

		__syncthreads();
	}

	if (idx == 0)
	{
		//unsigned int bound = g_bound[0];
		histogram_t* arr = s_Hist + pout*BinCount;
		
		//do binary search in CDF
		unsigned int beg = 0;
		unsigned int end = BinCount;
		while (beg < end)
		{
			unsigned int mid = beg + ((end - beg) >> 1);
			if (bound > arr[mid]) beg = mid + 1; else end = mid;
		}
		
		//g_dataMin[0] = dataMin + float(beg) * (dataMax - dataMin) / BinCount;
		//g_dataMax[0] = dataMin + float(beg+1) * (dataMax - dataMin) / BinCount;

		//assert( arr[beg] >= bound );
		g_boundParams[0] = bound - ((beg > 0) ? arr[beg-1] : 0);
		g_boundParams[1] = beg;
	}
}

SYNC_KERNEL_BEG(HISTOGRAM_WARPS_PER_BLOCK, histogramKernel, unsigned int _threadCount,
	const float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms
)
	const unsigned int BinCount = HISTOGRAM_BIN_COUNT;
	__shared__ histogram_t s_Hist[BinCount * WarpsPerBlock];

	histogram1<WarpsPerBlock, BlockSize, BinCount>(_threadCount, g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist);

	BLOCK_SYNC_BEGIN()

	histogram2<WarpsPerBlock, BlockSize, BinCount>(g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist, gridDim.x);

	BLOCK_SYNC_END()

SYNC_KERNEL_END()

BOUND_KERNEL_BEG(HISTOGRAM_WARPS_PER_BLOCK, histogram1Kernel,
	float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms
)
	const unsigned int BinCount = HISTOGRAM_BIN_COUNT;
	__shared__ histogram_t s_Hist[BinCount * WarpsPerBlock];

	histogram1<WarpsPerBlock, BlockSize, BinCount>(_threadCount, g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(HISTOGRAM_WARPS_PER_BLOCK, histogram2Kernel,
	float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms,
	unsigned int gridSize
)
	const unsigned int BinCount = HISTOGRAM_BIN_COUNT;
	__shared__ histogram_t s_Hist[BinCount * WarpsPerBlock];

	histogram2<WarpsPerBlock, BlockSize, BinCount>(g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist, gridSize);

BOUND_KERNEL_END()
