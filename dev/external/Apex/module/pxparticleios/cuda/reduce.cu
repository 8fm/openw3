#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.

#include "blocksync.cuh"
#include "reduce.cuh"

#include "include/common.h"
#include "include/reduce.h"


template <int WarpsPerBlock, int BlockSize>
inline __device__ void reduce1(unsigned int count, float* g_benefit, float4* g_output,
	unsigned int			*g_tmpActiveCount,
	float					*g_tmpBenefitSum,
	float					*g_tmpBenefitMin,
	float					*g_tmpBenefitMax,
	volatile unsigned int	*sdataActiveCount,
	volatile float			*sdataBenefitSum,
	volatile float			*sdataBenefitMin,
	volatile float			*sdataBenefitMax)
{
	unsigned int idx = threadIdx.x;

	sdataActiveCount[idx] = AddOPu::identity();
	sdataBenefitSum[idx]  = AddOPf::identity();
	sdataBenefitMin[idx]  = MinOPf::identity();
	sdataBenefitMax[idx]  = MaxOPf::identity();

	for (unsigned int pos = BlockSize*blockIdx.x + idx; pos < count; pos += BlockSize*gridDim.x)
	{
		float benefit = g_benefit[pos];
		if (benefit != -FLT_MAX)
		{
			sdataActiveCount[idx] = AddOPu::apply(sdataActiveCount[idx], 1);
			sdataBenefitSum[idx]  = AddOPf::apply(sdataBenefitSum[idx], benefit);
			sdataBenefitMin[idx]  = MinOPf::apply(sdataBenefitMin[idx], benefit);
			sdataBenefitMax[idx]  = MaxOPf::apply(sdataBenefitMax[idx], benefit);
		}
	}

	//don't need to synch because we use whole WARPs here
	reduceWarp<unsigned int, AddOPu>(sdataActiveCount);
	reduceWarp<float, AddOPf>(sdataBenefitSum);
	reduceWarp<float, MinOPf>(sdataBenefitMin);
	reduceWarp<float, MaxOPf>(sdataBenefitMax);

	//merge all warps for block
	__syncthreads();

	reduceBlock<unsigned int, AddOPu, WarpsPerBlock>(sdataActiveCount, g_tmpActiveCount);
	reduceBlock<float, AddOPf, WarpsPerBlock>(sdataBenefitSum,  g_tmpBenefitSum);
	reduceBlock<float, MinOPf, WarpsPerBlock>(sdataBenefitMin,  g_tmpBenefitMin);
	reduceBlock<float, MaxOPf, WarpsPerBlock>(sdataBenefitMax,  g_tmpBenefitMax);

	if (idx == 0) {
		__threadfence(); //only one write per block
	}
}

template <int WarpsPerBlock, int BlockSize>
inline __device__ void reduce2(float* g_benefit, float4* g_output,
	unsigned int			*g_tmpActiveCount,
	float					*g_tmpBenefitSum,
	float					*g_tmpBenefitMin,
	float					*g_tmpBenefitMax,
	volatile unsigned int	*sdataActiveCount,
	volatile float			*sdataBenefitSum,
	volatile float			*sdataBenefitMin,
	volatile float			*sdataBenefitMax,
	unsigned int gridSize)
{
	reduceGrid<unsigned int, AddOPu>(sdataActiveCount, g_tmpActiveCount, gridSize);
	reduceGrid<float, AddOPf>(sdataBenefitSum,  g_tmpBenefitSum, gridSize);
	reduceGrid<float, MinOPf>(sdataBenefitMin,  g_tmpBenefitMin, gridSize);
	reduceGrid<float, MaxOPf>(sdataBenefitMax,  g_tmpBenefitMax, gridSize);

	if (threadIdx.x == 0)
	{
		g_output[0] = make_float4(__int_as_float( sdataActiveCount[0] ),
			sdataBenefitSum[0],
			sdataBenefitMin[0],
			sdataBenefitMax[0]
		);
	}
}

#define REDUCE_KERNEL_SETUP() \
	unsigned int* g_tmpActiveCount = g_tmp; \
	float* g_tmpBenefitSum = (float*)(g_tmp + WARP_SIZE); \
	float* g_tmpBenefitMin = (float*)(g_tmp + WARP_SIZE*2); \
	float* g_tmpBenefitMax = (float*)(g_tmp + WARP_SIZE*3); \
	__shared__ volatile unsigned int	sdataActiveCount[BlockSize]; \
	__shared__ volatile float			sdataBenefitSum[BlockSize]; \
	__shared__ volatile float			sdataBenefitMin[BlockSize]; \
	__shared__ volatile float			sdataBenefitMax[BlockSize];

SYNC_KERNEL_BEG(REDUCE_WARPS_PER_BLOCK, reduceKernel,
	unsigned int count, float* g_benefit,
	float4* g_output, unsigned int* g_tmp
)
	REDUCE_KERNEL_SETUP()

	reduce1<WarpsPerBlock, BlockSize>(count, g_benefit, g_output,
		g_tmpActiveCount, g_tmpBenefitSum, g_tmpBenefitMin, g_tmpBenefitMax,
		sdataActiveCount, sdataBenefitSum, sdataBenefitMin, sdataBenefitMax);

	BLOCK_SYNC_BEGIN()

	reduce2<WarpsPerBlock, BlockSize>(g_benefit, g_output,
		g_tmpActiveCount, g_tmpBenefitSum, g_tmpBenefitMin, g_tmpBenefitMax,
		sdataActiveCount, sdataBenefitSum, sdataBenefitMin, sdataBenefitMax,
		gridDim.x);

	BLOCK_SYNC_END()

SYNC_KERNEL_END()

BOUND_KERNEL_BEG(REDUCE_WARPS_PER_BLOCK, reduce1Kernel,
	float* g_benefit, float4* g_output, unsigned int* g_tmp
)
	REDUCE_KERNEL_SETUP()

	reduce1<WarpsPerBlock, BlockSize>(_threadCount, g_benefit, g_output,
		g_tmpActiveCount, g_tmpBenefitSum, g_tmpBenefitMin, g_tmpBenefitMax,
		sdataActiveCount, sdataBenefitSum, sdataBenefitMin, sdataBenefitMax);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(REDUCE_WARPS_PER_BLOCK, reduce2Kernel,
	float* g_benefit, float4* g_output, unsigned int* g_tmp,
	unsigned int gridSize
)
	REDUCE_KERNEL_SETUP()

	reduce2<WarpsPerBlock, BlockSize>(g_benefit, g_output,
		g_tmpActiveCount, g_tmpBenefitSum, g_tmpBenefitMin, g_tmpBenefitMax,
		sdataActiveCount, sdataBenefitSum, sdataBenefitMin, sdataBenefitMax,
		gridSize);

BOUND_KERNEL_END()


