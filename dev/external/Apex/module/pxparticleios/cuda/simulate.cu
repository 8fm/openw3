#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.

#include "blocksync.cuh"
#include "common.cuh"

#include "include/common.h"
using namespace physx::apex;
using namespace physx::apex::pxparticleios;
#include "../include/ParticleIosCommon.h"
#include "include/simulate.h"

#include <ApexRand.h>
#include "hip/device_functions.h"

const unsigned int TAG_SHIFT = (32 - LOG2_WARP_SIZE);
const unsigned int TAG_MASK = (1U << TAG_SHIFT) - 1;

typedef volatile unsigned int histogram_t;

inline __device__ void addToBin(histogram_t *s_WarpHist, unsigned int data, unsigned int threadTag)
{
	unsigned int count;
	do {
		count = s_WarpHist[data] & TAG_MASK;
		count = threadTag | (count + 1);
		s_WarpHist[data] = count;
	} while (s_WarpHist[data] != count);
}

template <unsigned int BlockSize, unsigned int WarpsPerBlock, typename FieldAccessor>
__device__ void simulate(unsigned int targetCount,
	unsigned int lastCount,
	float deltaTime,
	physx::PxVec3 eyePos,
	InplaceHandle<InjectorParamsArray> injectorParamsArrayHandle,
	unsigned int injectorCount,
	unsigned int* g_holeScanSum,
	unsigned int* g_inputIdToParticleIndex,
	unsigned int* g_moveCount,
	unsigned int* g_tmpHistogram, 
	unsigned int* g_InjectorsCounters, 
	float4* g_positionMass,
	float4* g_velocityLife,
	float4* g_collisionNormalFlags,
	unsigned int* g_userData,
	float* g_lifeSpan,
	float* g_lifeTime,
	float* g_density,
	unsigned int* g_injector,
	NiIofxActorID* g_iofxActorIDs,
	float* g_benefit,
	float4* g_pxPosition,
	float4* g_pxVelocity,
	float4* g_pxCollisionNormals,
	float*  g_pxDensity,
	unsigned int* g_pxFlags,
	FieldAccessor& fieldAccessor,
	GridDensityParams simParams
	)
{
	__shared__ histogram_t s_Hist[HISTOGRAM_SIMULATE_BIN_COUNT * WarpsPerBlock];
	__shared__ unsigned int moveCount;
	const unsigned int warpIdx = (threadIdx.x >> LOG2_WARP_SIZE);
	histogram_t* s_WarpHist = s_Hist + warpIdx * HISTOGRAM_SIMULATE_BIN_COUNT;
	const unsigned int tag = (threadIdx.x & (WARP_SIZE-1)) << TAG_SHIFT;

	if (threadIdx.x == 0) {
		moveCount			= g_moveCount[0];
	}

	const InjectorParamsArray* injectorParamsArray = injectorParamsArrayHandle.resolve( KERNEL_CONST_MEM(simulateConstMem) );
	const Px3InjectorParams* injectorParamsElems = injectorParamsArray->getElems( KERNEL_CONST_MEM(simulateConstMem) );

	if (injectorCount <= HISTOGRAM_SIMULATE_BIN_COUNT)
	{
		//zero warp histograms
		#pragma unroll
		for(unsigned int i = 0; i < (HISTOGRAM_SIMULATE_BIN_COUNT >> LOG2_WARP_SIZE); i++) {
			s_Hist[threadIdx.x + i * BlockSize] = 0;
		}
	}

	__syncthreads();

	unsigned int step	= BlockSize*gridDim.x;

	for (unsigned int dstIdx = BlockSize*blockIdx.x + threadIdx.x; dstIdx < targetCount; dstIdx += step)
	{
		unsigned int srcIdx = dstIdx;

		unsigned int holeScanSum	= g_holeScanSum[dstIdx];

		if ((holeScanSum & HOLE_SCAN_FLAG) != 0)
		{
			// we have a hole
			holeScanSum &= HOLE_SCAN_MASK; //remove hole flag
			holeScanSum -= 1; //inclusive -> exclusive

			srcIdx = tex1Dfetch(KERNEL_TEX_REF(MoveIndices), holeScanSum + moveCount);
		}
		bool isNewParticle = (srcIdx >= lastCount);

		unsigned int outInj;
		float benefit = simulateParticle(
			injectorParamsElems,
			deltaTime,
			eyePos,
			isNewParticle,
			srcIdx,
			dstIdx,
			g_positionMass,
			g_velocityLife,
			g_collisionNormalFlags,
			g_userData,
			g_iofxActorIDs,
			g_lifeSpan,
			g_lifeTime,
			g_density,
			g_injector,
		    g_pxPosition,
			g_pxVelocity,
			g_pxCollisionNormals,
			g_pxDensity,
			g_pxFlags,
			fieldAccessor, 
			outInj,
			simParams
		);

		g_benefit[dstIdx] = benefit;

		if (outInj < injectorCount)
		{
			if (injectorCount <= HISTOGRAM_SIMULATE_BIN_COUNT)
			{
				//update per warp histogram
				addToBin(s_WarpHist, outInj, tag);
			}
			else
			{
				atomicAdd(g_InjectorsCounters + outInj, 1);
			}
		}
	}

	__syncthreads();

	if (injectorCount <= HISTOGRAM_SIMULATE_BIN_COUNT)
	{
		//merge warp histograms & output to global memory
		for (unsigned int pos = threadIdx.x; pos < injectorCount; pos += BlockSize)
		{
			unsigned int sum = 0;
			for(unsigned int i = 0; i < WarpsPerBlock; i++)
			{
				sum += s_Hist[pos + i * HISTOGRAM_SIMULATE_BIN_COUNT] & TAG_MASK;
			}
			g_tmpHistogram[blockIdx.x * HISTOGRAM_SIMULATE_BIN_COUNT + pos] = sum;  
		}
	}
}

template <int WarpsPerBlock, int BlockSize>
inline __device__ void mergeHistogram(
	unsigned int* g_InjectorsCounters,	unsigned int* g_tmpHistogram, unsigned int gridSize, unsigned int injectorCount
)
{
	for (unsigned int pos = threadIdx.x; pos < injectorCount; pos += BlockSize)
	{
		unsigned int sum = 0;
		for (unsigned int i = 0; i < gridSize; ++i)
		{
			sum += g_tmpHistogram[i*HISTOGRAM_SIMULATE_BIN_COUNT + pos];
		}
		g_InjectorsCounters[pos] = sum;
	}
}


BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	simulateKernel,
	unsigned int lastCount,
	float deltaTime,
	physx::PxVec3 eyePos,
	InplaceHandle<InjectorParamsArray> injectorParamsArrayHandle,
	unsigned int injectorCount,
	unsigned int* g_holeScanSum,
	unsigned int* g_inputIdToParticleIndex,
	unsigned int* g_moveCount,
	unsigned int* g_tmpHistogram, 
	unsigned int* g_InjectorsCounters,  
	float4* g_positionMass,
	float4* g_velocityLife,
	float4* g_collisionNormalFlags,
	unsigned int* g_userData,
	float* g_lifeSpan,
	float* g_lifeTime,
	float* g_density,
	unsigned int* g_injector,
	NiIofxActorID* g_iofxActorIDs,
	float* g_benefit,
    float4* g_pxPosition,
	float4* g_pxVelocity,
	float4* g_pxCollisionNormals,
	float*  g_pxDensity,
	unsigned int* g_pxFlags,
	GridDensityParams simParams
	)

	class FieldAccessor
	{
	public:
		__device__ PX_INLINE void operator() (unsigned int srcIdx, physx::PxVec3& velocityDelta ) { }
	} fieldAccessor;

	simulate<BlockSize, WarpsPerBlock>(_threadCount,
		lastCount,
		deltaTime,
		eyePos,
		injectorParamsArrayHandle,
		injectorCount,
		g_holeScanSum,
		g_inputIdToParticleIndex,
		g_moveCount,
		g_tmpHistogram, 
		g_InjectorsCounters,
		g_positionMass,
		g_velocityLife,
		g_collisionNormalFlags,
		g_userData,
		g_lifeSpan,
		g_lifeTime,
		g_density,
		g_injector,
		g_iofxActorIDs,
		g_benefit,
	    g_pxPosition,
		g_pxVelocity,
		g_pxCollisionNormals,
		g_pxDensity,
		g_pxFlags,
		fieldAccessor,
		simParams
	);
BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	simulateApplyFieldKernel,
	unsigned int lastCount,
	float deltaTime,
	physx::PxVec3 eyePos,
	InplaceHandle<InjectorParamsArray> injectorParamsArrayHandle,
	unsigned int injectorCount, 
	unsigned int* g_holeScanSum,
	unsigned int* g_inputIdToParticleIndex,
	unsigned int* g_moveCount,
	unsigned int* g_tmpHistogram, 
	unsigned int* g_InjectorsCounters,  
	float4* g_positionMass,
	float4* g_velocityLife,
	float4* g_collisionNormalFlags,
	unsigned int* g_userData,
	float* g_lifeSpan,
	float* g_lifeTime,
	float* g_density,
	unsigned int* g_injector,
	NiIofxActorID* g_iofxActorIDs,
	float* g_benefit,
    float4* g_pxPosition,
	float4* g_pxVelocity,
	float4* g_pxCollisionNormals,
	float*  g_pxDensity,
	unsigned int* g_pxFlags,
	GridDensityParams simParams
	)

	class FieldAccessor
	{
	public:
		__device__ PX_INLINE void operator() (unsigned int srcIdx, physx::PxVec3& velocityDelta )
		{
			float4 field = tex1Dfetch(KERNEL_TEX_REF(Field), srcIdx);
			velocityDelta.x += field.x;
			velocityDelta.y += field.y;
			velocityDelta.z += field.z;
		}
	} fieldAccessor;

	simulate<BlockSize, WarpsPerBlock>(_threadCount,
		lastCount,
		deltaTime,
		eyePos,
		injectorParamsArrayHandle,
		injectorCount,
		g_holeScanSum,
		g_inputIdToParticleIndex,
		g_moveCount,
		g_tmpHistogram, 
		g_InjectorsCounters,  
		g_positionMass,
		g_velocityLife,
		g_collisionNormalFlags,
		g_userData,
		g_lifeSpan,
		g_lifeTime,
		g_density,
		g_injector,
		g_iofxActorIDs,
		g_benefit,
	    g_pxPosition,
		g_pxVelocity,
		g_pxCollisionNormals,
		g_pxDensity,
		g_pxFlags,
		fieldAccessor,
		simParams
	);
BOUND_KERNEL_END()


BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK, mergeHistogramKernel,
	unsigned int* g_InjectorsCounters, unsigned int* g_tmpHistogram, unsigned int gridSize, unsigned int injectorCount
)

	mergeHistogram<WarpsPerBlock, BlockSize>(g_InjectorsCounters, g_tmpHistogram, gridSize, injectorCount);

BOUND_KERNEL_END()


BOUND_KERNEL_BEG(STATE_WARPS_PER_BLOCK,
	stateKernel,
	unsigned int lastCount,
	unsigned int targetCount,
	unsigned int* g_moveCount,
	unsigned int* g_inStateToInput,
	unsigned int* g_outStateToInput)

	__shared__ unsigned int moveCount;
	if (threadIdx.x == 0) {
		moveCount = g_moveCount[0];
	}
	__syncthreads();

	for (unsigned int idx = BlockSize*blockIdx.x + threadIdx.x; idx < _threadCount; idx += BlockSize*gridDim.x)
	{
		unsigned int newInputIdx = NiIosBufferDesc::NOT_A_PARTICLE;

		unsigned int lastInputIdx = (idx < lastCount) ? g_outStateToInput[idx] : idx;

		unsigned int holeScanSum = tex1Dfetch(KERNEL_TEX_REF(HoleScanSum), lastInputIdx);
		if ((holeScanSum & HOLE_SCAN_FLAG) == 0)
		{
			//non-hole
			newInputIdx = lastInputIdx;
			if (lastInputIdx >= targetCount)
			{
				//inverse scan for non-holes:
				//holeIdx - inclusive scan of holes, and we need exclusive scan of non-holes
				//moveCount - count of holes before targetCount
				unsigned int moveIdx = (lastInputIdx - targetCount) - (holeScanSum - moveCount);
				newInputIdx = tex1Dfetch(KERNEL_TEX_REF(MoveIndices), moveIdx);
			}

			if (lastInputIdx >= lastCount)
			{
				//new particle
				newInputIdx |= NiIosBufferDesc::NEW_PARTICLE_FLAG;
			}
		}

		g_inStateToInput[idx] = newInputIdx;
	}
BOUND_KERNEL_END()


BOUND_KERNEL_BEG(STATE_WARPS_PER_BLOCK,
	testKernel,
	unsigned int scalarVar,
	unsigned int* vectorVar,
	InplaceHandle<int> multHandle)

	unsigned int idx = BlockSize*blockIdx.x + threadIdx.x;	
	if (idx < _threadCount)
	{
		int mult = *(multHandle.resolve(KERNEL_CONST_MEM(simulateConstMem)));
		vectorVar[idx] = vectorVar[idx] + scalarVar + tex1Dfetch(KERNEL_TEX_REF(HoleScanSum), _threadCount - idx - 1);
		vectorVar[idx] *= mult;
	}
BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	gridDensityGridClearKernel,
	float* gridDensityGrid,
	GridDensityParams simParams
	)
{
	unsigned int step = BlockSize*gridDim.x;
	for (unsigned int dstIdx = BlockSize*blockIdx.x + threadIdx.x; dstIdx < _threadCount; dstIdx += step)
	{
		gridDensityGrid[dstIdx] = 0.0f;
	}
}
BOUND_KERNEL_END()

inline __device__ unsigned int gridAddress3d(unsigned int c, unsigned int b, unsigned int a, unsigned int dim)
{
	return (c*dim + b)*dim + a;
}

BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	gridDensityGridFillKernel,
	float4* positionMass,
	float* gridDensityGrid,
	GridDensityParams simParams
	)
{
	unsigned int dim = simParams.GridResolution;
	unsigned int step = BlockSize*gridDim.x;
	for (unsigned int dstIdx = BlockSize*blockIdx.x + threadIdx.x; dstIdx < _threadCount; dstIdx += step)
	{
		unsigned int srcIdx = dstIdx;
		physx::PxVec3 position;
		splitFloat4(position, positionMass[srcIdx]);

		position -= simParams.DensityOrigin;
		float size = simParams.GridSize;
		position /= size;
		position.x += 0.5;
		position.y += 0.5;
		position.z += 0.5;
		// position now normalized to [0:1]
		int a = floor(dim*position.x);
		int b = floor(dim*position.y);
		int c = floor(dim*position.z);
		if( a >= 0 && a < dim &&
			b >= 0 && b < dim &&
			c >= 0 && c < dim)
		{
			//physx::PxU32 loc = c*dim*dim+b*dim+a;
			#if (__CUDA_ARCH__ >= 200)
			atomicAdd(&gridDensityGrid[gridAddress3d(c,b,a,dim)],1.0f);
			#endif
		}
	}
}
BOUND_KERNEL_END()

inline __device__ float safeGridRead(float* density,unsigned int dim,int c,int b,int a)
{
	if( a >= 0 && a < dim &&
		b >= 0 && b < dim &&
		c >= 0 && c < dim )
	{
		return density[gridAddress3d(c,b,a,dim)];
	}
	else
	{
		return 0.f;
	}
}

BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	gridDensityGridApplyKernel,
	float* density,
	float4* positionMass,
	float* gridDensityGrid,
	GridDensityParams simParams
	)
{
	unsigned int dim = simParams.GridResolution;
	unsigned int step = BlockSize*gridDim.x;
	for (unsigned int dstIdx = BlockSize*blockIdx.x + threadIdx.x; dstIdx < _threadCount; dstIdx += step)
	{
		unsigned int srcIdx = dstIdx;
		physx::PxVec3 position;
		splitFloat4(position, positionMass[srcIdx]);

		position -= simParams.DensityOrigin;
		float size = simParams.GridSize;
		position /= size;
		position.x += 0.5;
		position.y += 0.5;
		position.z += 0.5;
		// position now normalized to [0:1]
		float u = dim*position.x - 0.5f;
		float v = dim*position.y - 0.5f;
		float w = dim*position.z - 0.5f;
		float uf = u - floor(u);
		float vf = v - floor(v);
		float wf = w - floor(w);
		int ui = floor(u);
		int vi = floor(v);
		int wi = floor(w);
		const float v000 = safeGridRead(gridDensityGrid,dim,wi  ,vi  ,ui  );
		const float v100 = safeGridRead(gridDensityGrid,dim,wi  ,vi  ,ui+1);
		const float v010 = safeGridRead(gridDensityGrid,dim,wi  ,vi+1,ui  );
		const float v110 = safeGridRead(gridDensityGrid,dim,wi  ,vi+1,ui+1);
		const float v001 = safeGridRead(gridDensityGrid,dim,wi+1,vi  ,ui  );
		const float v101 = safeGridRead(gridDensityGrid,dim,wi+1,vi  ,ui+1);
		const float v011 = safeGridRead(gridDensityGrid,dim,wi+1,vi+1,ui  );
		const float v111 = safeGridRead(gridDensityGrid,dim,wi+1,vi+1,ui+1);
		const float c00 = v000*(1.f-uf)+v100*uf;
		const float c10 = v010*(1.f-uf)+v110*uf;
		const float c01 = v001*(1.f-uf)+v101*uf;
		const float c11 = v011*(1.f-uf)+v111*uf;
		const float c0 = c00*(1.f-vf)+c10*vf;
		const float c1 = c01*(1.f-vf)+c11*vf;
		const float c = c0*(1.f-wf)+c1*wf;
		if(density)
		{
			density[dstIdx] = c/simParams.GridMaxCellCount;
		}
	}
}
BOUND_KERNEL_END()

// ******************** Frustum Density **********************

BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	gridDensityGridFillFrustumKernel,
	float4* positionMass,
	float* gridDensityGrid,
	GridDensityParams simParams,
	::physx::PxMat44 mat,
	GridDensityFrustumParams frustum
	)
{
	unsigned int dim = simParams.GridResolution;
	unsigned int step = BlockSize*gridDim.x;
	for (unsigned int dstIdx = BlockSize*blockIdx.x + threadIdx.x; dstIdx < _threadCount; dstIdx += step)
	{
		unsigned int srcIdx = dstIdx;
		physx::PxVec3 position;
		splitFloat4(position, positionMass[srcIdx]);

		physx::PxVec4 pos(position.x,position.y,position.z,1.f);
		pos = mat.transform(pos);
		// find norm x and y
		position.x = pos.x/pos.w;
		position.y = pos.y/pos.w;
		position.z = min(sqrt(pos.z),0.999f); //pow(pos.z,frustum.gamma);
		// position now normalized to [0:1]
		int a = floor(dim*position.x);
		int b = floor(dim*position.y);
		int c = floor(dim*position.z);
		if( a >= 0 && a < dim &&
			b >= 0 && b < dim &&
			c >= 0 && c < dim)
		{
			// compute density value
			float x0 = frustum.nearDimX*(1.f-pos.z)+frustum.farDimX*(pos.z);
			float y0 = frustum.nearDimY*(1.f-pos.z)+frustum.farDimY*(pos.z);
			float z0 = frustum.dimZ;
			float k = dim*dim*dim/(x0*y0*z0);
			#if (__CUDA_ARCH__ >= 200)
			atomicAdd(&gridDensityGrid[gridAddress3d(c,b,a,dim)],k);
			#endif
		}
	}
}
BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	gridDensityGridApplyFrustumKernel,
	float* density,
	float4* positionMass,
	float* gridDensityGrid,
	GridDensityParams simParams,
	::physx::PxMat44 mat,
	GridDensityFrustumParams frustum
	)
{
	unsigned int dim = simParams.GridResolution;
	unsigned int step = BlockSize*gridDim.x;
	for (unsigned int dstIdx = BlockSize*blockIdx.x + threadIdx.x; dstIdx < _threadCount; dstIdx += step)
	{
		unsigned int srcIdx = dstIdx;
		physx::PxVec3 position;
		splitFloat4(position, positionMass[srcIdx]);

		physx::PxVec4 pos(position.x,position.y,position.z,1.f);
		pos = mat.transform(pos);
		// find norm x and y
		position.x = pos.x/pos.w;
		position.y = pos.y/pos.w;
		position.z = min(sqrt(pos.z),0.999f); //pow(pos.z,frustum.gamma);
		// position now normalized to [0:1]
		float u = dim*position.x- 0.5f;
		float v = dim*position.y- 0.5f;
		float w = dim*position.z- 0.5f;
		float uf = u - floor(u);
		float vf = v - floor(v);
		float wf = w - floor(w);
		int ui = floor(u);
		int vi = floor(v);
		int wi = floor(w);
		const float v000 = safeGridRead(gridDensityGrid,dim,wi  ,vi  ,ui  );
		const float v100 = safeGridRead(gridDensityGrid,dim,wi  ,vi  ,ui+1);
		const float v010 = safeGridRead(gridDensityGrid,dim,wi  ,vi+1,ui  );
		const float v110 = safeGridRead(gridDensityGrid,dim,wi  ,vi+1,ui+1);
		const float v001 = safeGridRead(gridDensityGrid,dim,wi+1,vi  ,ui  );
		const float v101 = safeGridRead(gridDensityGrid,dim,wi+1,vi  ,ui+1);
		const float v011 = safeGridRead(gridDensityGrid,dim,wi+1,vi+1,ui  );
		const float v111 = safeGridRead(gridDensityGrid,dim,wi+1,vi+1,ui+1);
		const float c00 = v000*(1.f-uf)+v100*uf;
		const float c10 = v010*(1.f-uf)+v110*uf;
		const float c01 = v001*(1.f-uf)+v101*uf;
		const float c11 = v011*(1.f-uf)+v111*uf;
		const float c0 = c00*(1.f-vf)+c10*vf;
		const float c1 = c01*(1.f-vf)+c11*vf;
		const float c = c0*(1.f-wf)+c1*wf;
		if(density)
		{
			density[dstIdx] = min(max(c/simParams.GridMaxCellCount,0.f),1.1f);
		}
	}
}
BOUND_KERNEL_END()

// ************* LowPass*******************
FREE_KERNEL_BEG(SIMULATE_WARPS_PER_BLOCK,
	gridDensityGridLowPassKernel,
	float* gridDensityGridIn,
	float* gridDensityGridOut,
	GridDensityParams simParams
	)
{
	unsigned int dim = simParams.GridResolution;
	unsigned int step = BlockSize*gridDim.x;
	for (unsigned int dstIdx = BlockSize*blockIdx.x + threadIdx.x; dstIdx < _threadCount; dstIdx += step)
	{
		int a = dstIdx%dim;
		int b = (dstIdx/dim)%dim;
		int c = (dstIdx/(dim*dim))%dim;
		float value = 0.f;

		// Just a box filter for now...

		// neg z
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b-1,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b-1,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b-1,a+1);
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b+0,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b+0,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b+0,a+1);
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b+1,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b+1,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c-1,b+1,a+1);
		// zero z
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b-1,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b-1,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b-1,a+1);
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b+0,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b+0,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b+0,a+1);
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b+1,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b+1,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+0,b+1,a+1);
		// pos z
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b-1,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b-1,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b-1,a+1);
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b+0,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b+0,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b+0,a+1);
		value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b+1,a-1); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b+1,a+0); value += 1.0f*safeGridRead(gridDensityGridIn,dim,c+1,b+1,a+1);

		gridDensityGridOut[gridAddress3d(c,b,a,dim)] = value/27.f;
	}
}
FREE_KERNEL_END()