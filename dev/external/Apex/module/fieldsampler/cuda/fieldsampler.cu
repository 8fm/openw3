#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.

#include "common.cuh"

#include "include/common.h"
using namespace physx::apex;
#include "include/fieldsampler.h"
#include "../include/FieldSamplerCommon.h"


BOUND_KERNEL_BEG(CLEAR_WARPS_PER_BLOCK, clearKernel,
	float4* g_accumField, float4* g_accumVelocity
)
	for (unsigned int idx = BlockSize*blockIdx.x + threadIdx.x; idx < _threadCount; idx += BlockSize*gridDim.x)
	{
		g_accumField[idx] = make_float4(0, 0, 0, 0);
		g_accumVelocity[idx] = make_float4(0, 0, 0, 0);
	}
BOUND_KERNEL_END()


BOUND_KERNEL_BEG(COMPOSE_WARPS_PER_BLOCK, composeKernel,
	float4* g_accumField, const float4* g_accumVelocity, const float4* g_velocity, physx::PxF32 timestep
)
	for (unsigned int idx = BlockSize*blockIdx.x + threadIdx.x; idx < _threadCount; idx += BlockSize*gridDim.x)
	{
		float4 avel4 = g_accumVelocity[idx];
		physx::PxVec3 avel(avel4.x, avel4.y, avel4.z);
		physx::PxF32 avelW = avel4.w;

		if (avelW >= VELOCITY_WEIGHT_THRESHOLD)
		{
			float4 vel4 = g_velocity[idx];
			physx::PxVec3 vel(vel4.x, vel4.y, vel4.z);

			float4 field4 = g_accumField[idx];
			physx::PxVec3 field(field4.x, field4.y, field4.z);

			field += (avel - avelW * vel);

			g_accumField[idx] = make_float4(field.x, field.y, field.z, 0);
		}
	}
BOUND_KERNEL_END()


BOUND_KERNEL_BEG(CLEAR_GRID_WARPS_PER_BLOCK, clearGridKernel,
	physx::PxU32 numX, physx::PxU32 numY, physx::PxU32 numZ,
	physx::PxU32 strideX, physx::PxU32 strideY,
	float4* g_accumVelocity
)
	for (unsigned int ithread = BlockSize*blockIdx.x + threadIdx.x; ithread < _threadCount; ithread += BlockSize*gridDim.x)
	{
		int ixy = (ithread / strideY);
		int iz  = (ithread % strideY);
		int iy  = (ixy % numY);
		int ix  = (ixy / numY);

		int idx = (ix * strideX) + (iy * strideY) + iz;

		if (ix < numX && iy < numY && iz < numZ)
		{
			g_accumVelocity[idx] = make_float4(0, 0, 0, 0);
		}
	}
BOUND_KERNEL_END()

BOUND_KERNEL_BEG(APPLY_PARTICLES_WARPS_PER_BLOCK, applyParticlesKernel,
	float4* g_velocity, const float4* g_outField
)
	for (unsigned int idx = BlockSize*blockIdx.x + threadIdx.x; idx < _threadCount; idx += BlockSize*gridDim.x)
	{
		g_velocity[idx].x += g_outField[idx].x;
		g_velocity[idx].y += g_outField[idx].y;
		g_velocity[idx].z += g_outField[idx].z;
	}
BOUND_KERNEL_END()

#ifdef APEX_TEST

BOUND_KERNEL_BEG(TEST_PARTICLES_WARPS_PER_BLOCK, testParticleKernel,
	float4* g_position, float4* g_velocity,
	physx::PxU32* g_flag,
	const float4* g_initPosition, const float4* g_initVelocity
)
	for (unsigned int idx = BlockSize*blockIdx.x + threadIdx.x; idx < _threadCount; idx += BlockSize*gridDim.x)
	{
		
		testParticle((physx::PxVec4&)g_position[idx], (physx::PxVec4&)g_velocity[idx], g_flag[idx], (physx::PxVec4&)g_initPosition[idx], (physx::PxVec4&)g_initVelocity[idx]);
	}
BOUND_KERNEL_END()

#endif