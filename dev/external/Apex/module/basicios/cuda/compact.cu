#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.


#include "common.cuh"

#include "include/common.h"
#include "include/compact.h"


inline __device__ unsigned int fetchHoleScan(unsigned int index, unsigned int& flag)
{
	const unsigned int holeScan = tex1Dfetch(KERNEL_TEX_REF(CompactScanSum), index);
	flag = (holeScan >> HOLE_SCAN_FLAG_BIT);
	return (holeScan & HOLE_SCAN_MASK); //inclusive
}
inline __device__ unsigned int fetchOutputScan(unsigned int index, unsigned int start, unsigned int holesBeforeStart, unsigned int& flag)
{
	unsigned int holeScan = fetchHoleScan(index, flag);
	if (index < start)
	{
		return holeScan; //inclusive
	}
	else
	{
		flag ^= 1;
		const unsigned int nonHoleScan = (index + 1) - holeScan; //inclusive
		const unsigned int nonHolesBeforeStart = start - holesBeforeStart;
		return holesBeforeStart + (nonHoleScan - nonHolesBeforeStart); //inclusive
	}
}

BOUND_KERNEL_BEG(COMPACT_WARPS_PER_BLOCK, compactKernel,
	unsigned int targetCount, unsigned int totalCount, unsigned int injectorCount, unsigned int* g_outIndices, unsigned int* g_outCount, unsigned int* g_injCounters
)
	const unsigned int start = targetCount;

	__shared__ unsigned int holesBeforeStart;
	if (threadIdx.x == 0) {
		unsigned int flag;
		holesBeforeStart = (start > 0) ? fetchHoleScan(start-1, flag) : 0;
		if (blockIdx.x == 0) {
			g_outCount[0] = holesBeforeStart;
		}
	}
	__syncthreads();

	{
		const unsigned int idx = threadIdx.x;
		const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
		const unsigned int idxInWarp = idx & (WARP_SIZE-1);

		const unsigned int CountPerBlock = (totalCount + gridDim.x-1) / gridDim.x;

		const unsigned int DataWarpsPerBlock = (CountPerBlock + WARP_SIZE-1) / WARP_SIZE;
		const unsigned int WarpBorder = DataWarpsPerBlock % WarpsPerBlock;
		const unsigned int WarpFactor = DataWarpsPerBlock / WarpsPerBlock;

		const unsigned int WarpSelect = (warpIdx < WarpBorder) ? 1 : 0;
		const unsigned int WarpCount = WarpFactor + WarpSelect;
		const unsigned int WarpOffset = warpIdx * WarpCount + WarpBorder * (1 - WarpSelect);

		const unsigned int blockBeg = blockIdx.x * CountPerBlock;
		const unsigned int blockEnd = min(blockBeg + CountPerBlock, totalCount);

		const unsigned int warpBeg = blockBeg + (WarpOffset << LOG2_WARP_SIZE);
		const unsigned int warpEnd = min(warpBeg + (WarpCount << LOG2_WARP_SIZE), blockEnd);

		const unsigned int Log2BufferSize = (LOG2_WARP_SIZE + 1);
		const unsigned int BufferSize = (1 << Log2BufferSize);

		__shared__ volatile unsigned int sdata[WARP_SIZE * WarpsPerBlock];

		__shared__ volatile unsigned int buffer[BufferSize*WarpsPerBlock];
		__shared__ volatile unsigned int outputBeg[WarpsPerBlock];

		if (warpBeg < warpEnd)
		{
			if (idxInWarp == 0) {
				unsigned int flag;
				outputBeg[warpIdx] = (warpBeg > 0) ? fetchOutputScan(warpBeg-1, start, holesBeforeStart, flag) : 0;
			}

			unsigned int bufferBeg = outputBeg[warpIdx] & (WARP_SIZE-1);
			unsigned int bufferEnd = bufferBeg;

			if (idxInWarp == 0) {
				outputBeg[warpIdx] &= ~(WARP_SIZE-1);
			}

			for (unsigned int i = warpBeg; i < warpEnd; i += WARP_SIZE)
			{
				unsigned int inputPos = i + idxInWarp;
				if (inputPos < warpEnd)
				{
					unsigned int flag;
					unsigned int outputPos = fetchOutputScan(inputPos, start, holesBeforeStart, flag);
					unsigned int bufferPos = outputPos - outputBeg[warpIdx];

					sdata[idx] = bufferPos;
					if (flag)
					{
						bufferPos -= 1; //inclusive -> exclusive
						bufferPos += (bufferBeg & WARP_SIZE);
						bufferPos &= (BufferSize-1);

						buffer[(warpIdx << Log2BufferSize) + bufferPos] = inputPos;
					}
				}

				unsigned int endOfWarp = (min(i + WARP_SIZE, warpEnd)-1 - warpBeg) & (WARP_SIZE-1);
				bufferEnd = (bufferBeg & WARP_SIZE) + sdata[(warpIdx << LOG2_WARP_SIZE) + endOfWarp];
				bufferEnd &= (BufferSize-1);

				if ((bufferBeg & WARP_SIZE) != (bufferEnd & WARP_SIZE))
				{
					if (idxInWarp >= (bufferBeg & (WARP_SIZE-1)) ) {
						g_outIndices[outputBeg[warpIdx] + idxInWarp] = buffer[(warpIdx << Log2BufferSize) + (bufferBeg & WARP_SIZE) + idxInWarp];
					}
					bufferBeg = (bufferEnd & WARP_SIZE);
					if (idxInWarp == 0) {
						outputBeg[warpIdx] += WARP_SIZE;
					}
				}
			}

			if ( idxInWarp >= (bufferBeg & (WARP_SIZE-1)) && idxInWarp < (bufferEnd & (WARP_SIZE-1)) ) {
				g_outIndices[outputBeg[warpIdx] + idxInWarp] = buffer[(warpIdx << Log2BufferSize) + (bufferBeg & WARP_SIZE) + idxInWarp];
			}
		}
	}

	if (injectorCount > HISTOGRAM_SIMULATE_BIN_COUNT)
	{
		for (physx::PxU32 pos = BlockSize*blockIdx.x + threadIdx.x; pos < injectorCount; pos += BlockSize*gridDim.x)
		{
			g_injCounters[ pos ] = 0;
		}
	}
BOUND_KERNEL_END()
