#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2013 NVIDIA Corporation. All rights reserved.

#define FIX_FOR_KEPLER 1

#include "blocksync.cuh"

#include "include/common.h"
#include "include/scan.h"

inline __device__ void reduceWarp(unsigned int idx, volatile unsigned int* sdata)
{
	sdata[idx] += sdata[idx + 16];
	sdata[idx] += sdata[idx +  8];
	sdata[idx] += sdata[idx +  4];
	sdata[idx] += sdata[idx +  2];
	sdata[idx] += sdata[idx +  1];
}

inline __device__ void bitScanWarp(unsigned int idx, unsigned idxInWarp, volatile unsigned int* sdata)
{
	sdata[idx] <<= idxInWarp;
	reduceWarp(idx, sdata);

	unsigned int result = sdata[ idx & ~(WARP_SIZE-1) ];

	sdata[idx] = __popc( result & (0xFFFFFFFFU >> (31-idxInWarp)) );
}

inline __device__ int evalBin(float benefit, float benefitMin, float benefitMax)
{
	return (benefit > -FLT_MAX) ? ((benefit - benefitMin) * HISTOGRAM_BIN_COUNT / (benefitMax - benefitMin)) : -1;
}

inline __device__ unsigned int condition(int bin, int boundBin)
{
	return (bin < boundBin) ? 1 : 0;
}
inline __device__ unsigned int condition1(int bin, int boundBin)
{
	return (bin == boundBin) ? 1 : 0;
}


#define SCAN_LOOP_COMMON(whole) \
	sdata[idx] = marker; \
	bitScanWarp(idx, idxInWarp, sdata); \
	if (whole || pos < warpEnd) g_indices[pos] = (prefix + sdata[idx]) | (marker << HOLE_SCAN_FLAG_BIT); \
	if (whole) prefix += sdata[(warpIdx << LOG2_WARP_SIZE) + WARP_SIZE-1];

#define SCAN_LOOP_1(whole) \
	unsigned int marker = 0; \
	if (whole || pos < warpEnd) { \
		float benefit = g_benefits[pos]; \
		int bin = evalBin(benefit, benefitMin, benefitMax); \
		marker = condition(bin, markBin); \
	} \
	SCAN_LOOP_COMMON(whole)

#define SCAN_LOOP_2(whole) \
	unsigned int marker = 0; \
	if (whole || pos < warpEnd) { \
		float benefit = g_benefits[pos]; \
		int bin = evalBin(benefit, benefitMin, benefitMax); \
		marker = condition(bin, boundBin); \
		marker |= condition1(bin, boundBin) << 1; \
	} \
	if (prefix1 < boundCount) \
	{ \
		sdata1[idx] = (marker >> 1); \
		bitScanWarp(idx, idxInWarp, sdata1); \
		marker |= (marker >> 1) & ((prefix1 + sdata1[idx] <= boundCount) ? 1 : 0); \
		if (whole) prefix1 += sdata1[(warpIdx << LOG2_WARP_SIZE) + WARP_SIZE-1]; \
	} \
	marker &= 1; \
	SCAN_LOOP_COMMON(whole)


template <int WarpsPerBlock, int BlockSize>
inline __device__ void scan1(unsigned int count,
	float benefitMin, float benefitMax, unsigned int* g_indices, const float* g_benefits,
	unsigned int* g_boundParams, unsigned int* g_tmpCounts, unsigned int* g_tmpCounts1,
	volatile unsigned int* sdata, volatile unsigned int* sdata1,
	const unsigned int warpBeg, const unsigned int warpEnd)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);

	__shared__ int boundBin;
	if (idx == 0)
	{
		boundBin = g_boundParams[1];
	}
	__syncthreads();

	sdata[idx] = 0;
	sdata1[idx] = 0;

	if (warpBeg < warpEnd)
	{
		//accum
		for (unsigned int i = warpBeg + idxInWarp; i < warpEnd; i += WARP_SIZE)
		{
			float benefit = g_benefits[i];

			int bin = evalBin(benefit, benefitMin, benefitMax);
			sdata[idx] += condition(bin, boundBin);
			sdata1[idx] += condition1(bin, boundBin);
		}
		//reduce warp
		reduceWarp(idx, sdata);
		reduceWarp(idx, sdata1);
	}

	__syncthreads();

	if (idx < WarpsPerBlock)
	{
		g_tmpCounts[blockIdx.x * WarpsPerBlock + idx] = sdata[idx << LOG2_WARP_SIZE];
		g_tmpCounts1[blockIdx.x * WarpsPerBlock + idx] = sdata1[idx << LOG2_WARP_SIZE];
		__threadfence();
	}
}


inline __device__ void scanWarp(unsigned int scanIdx, volatile unsigned int* sdata)
{
	sdata[scanIdx] += sdata[scanIdx -  1];
	sdata[scanIdx] += sdata[scanIdx -  2];
	sdata[scanIdx] += sdata[scanIdx -  4];
	sdata[scanIdx] += sdata[scanIdx -  8];
	sdata[scanIdx] += sdata[scanIdx - 16]; 
}


template <int WarpsPerBlock, int BlockSize>
inline __device__ void scan2(
	float benefitMin, float benefitMax, unsigned int* g_indices, const float* g_benefits,
	unsigned int* g_boundParams, unsigned int* g_tmpCounts, unsigned int* g_tmpCounts1,
	volatile unsigned int* sdata, volatile unsigned int* sdata1,
	unsigned int gridSize)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int WarpsPerGrid = WarpsPerBlock * gridSize;

#if FIX_FOR_KEPLER
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);

	unsigned int val = (idx < WarpsPerGrid) ? g_tmpCounts[idx] : 0;
	unsigned int val1 = (idx < WarpsPerGrid) ? g_tmpCounts1[idx] : 0;

	//setup scan
	int scanIdx = (warpIdx << (LOG2_WARP_SIZE + 1)) + idxInWarp;
	sdata[scanIdx] = 0;
	sdata1[scanIdx] = 0;
	scanIdx += WARP_SIZE;
	sdata[scanIdx] = val;
	sdata1[scanIdx] = val1;

	scanWarp(scanIdx, sdata);
	scanWarp(scanIdx, sdata1);

	//__syncthreads();

	val = sdata[scanIdx];
	val1 = sdata1[scanIdx];

	__syncthreads();

	if (idxInWarp == WARP_SIZE-1)
	{
		sdata[warpIdx + WARP_SIZE] = val;
		sdata1[warpIdx + WARP_SIZE] = val1;
	}
	__syncthreads();

	if (idx < WARP_SIZE)
	{
		scanWarp(scanIdx, sdata);
		scanWarp(scanIdx, sdata1);
	}
	__syncthreads();

	val += sdata[warpIdx + WARP_SIZE - 1];
	val1 += sdata1[warpIdx + WARP_SIZE - 1];

	if (idx < WarpsPerGrid) {
		g_tmpCounts[idx] = val;
		g_tmpCounts1[idx] = val1;
		__threadfence();
	}
#else
	//do prefix sum (TODO: optimize for warps)
	sdata[idx] = (idx < WarpsPerGrid) ? g_tmpCounts[idx] : 0;
	sdata1[idx] = (idx < WarpsPerGrid) ? g_tmpCounts1[idx] : 0;

	int pout = 0;
	int pin = 1;

	#pragma unroll
	for (int offset = 1; offset < BlockSize; offset *= 2)
	{
		pout = 1 - pout;
		pin  = 1 - pout;

		sdata[pout*BlockSize + idx] = sdata[pin*BlockSize + idx];
		if (idx >= offset) sdata[pout*BlockSize + idx] += sdata[pin*BlockSize + idx - offset];

		sdata1[pout*BlockSize + idx] = sdata1[pin*BlockSize + idx];
		if (idx >= offset) sdata1[pout*BlockSize + idx] += sdata1[pin*BlockSize + idx - offset];

		__syncthreads();
	}

	if (idx < WarpsPerGrid) {
		g_tmpCounts[idx] = sdata[pout*BlockSize + idx];
		g_tmpCounts1[idx] = sdata1[pout*BlockSize + idx];
		__threadfence();
	}
#endif
}

template <int WarpsPerBlock, int BlockSize>
inline __device__ void scan3(unsigned int count,
	float benefitMin, float benefitMax, unsigned int* g_indices, const float* g_benefits,
	unsigned int* g_boundParams, unsigned int* g_tmpCounts, unsigned int* g_tmpCounts1,
	volatile unsigned int* sdata, volatile unsigned int* sdata1,
	const unsigned int warpBeg, const unsigned int warpEnd)
{
	const unsigned int idx = threadIdx.x;
	const unsigned int idxInWarp = idx & (WARP_SIZE-1);
	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);

	__shared__ unsigned int sCounts[WarpsPerBlock+1];
	__shared__ unsigned int sCounts1[WarpsPerBlock+1];

	__shared__ unsigned int boundCount;
	__shared__ int          boundBin;
	if (idx == 0)
	{
		boundCount = g_boundParams[0];
		boundBin   = g_boundParams[1];

		sCounts[0]  = (blockIdx.x > 0) ? g_tmpCounts[blockIdx.x * WarpsPerBlock - 1] : 0;
		sCounts1[0] = (blockIdx.x > 0) ? g_tmpCounts1[blockIdx.x * WarpsPerBlock - 1] : 0;
	}
	if (idx < WarpsPerBlock)
	{
		sCounts[idx+1]  = g_tmpCounts[blockIdx.x * WarpsPerBlock + idx];
		sCounts1[idx+1] = g_tmpCounts1[blockIdx.x * WarpsPerBlock + idx];
	}
	__syncthreads();

	if (warpBeg < warpEnd)
	{
		unsigned int prefix = sCounts[warpIdx];
		unsigned int prefix1 = sCounts1[warpIdx];

		if (prefix1 >= boundCount || boundCount >= sCounts1[warpIdx+1])
		{
			prefix += min(prefix1, boundCount);
			int markBin = (prefix1 >= boundCount) ? boundBin : (boundBin + 1);

			unsigned int pos;
			for (pos = warpBeg + idxInWarp; pos < (warpEnd & ~(WARP_SIZE-1)); pos += WARP_SIZE)
			{
				SCAN_LOOP_1(true)
			}
			if ((warpEnd & (WARP_SIZE-1)) > 0)
			{
				SCAN_LOOP_1(false)
			}
		}
		else
		{
			prefix += prefix1;

			unsigned int pos;
			for (pos = warpBeg + idxInWarp; pos < (warpEnd & ~(WARP_SIZE-1)); pos += WARP_SIZE)
			{
				SCAN_LOOP_2(true)
			}
			if ((warpEnd & (WARP_SIZE-1)) > 0)
			{
				SCAN_LOOP_2(false)
			}
		}
	}
}

#define SCAN_KERNEL_SETUP(count) \
	const unsigned int DataWarpsPerGrid = ((count + WARP_SIZE-1) >> LOG2_WARP_SIZE); \
	const unsigned int DataWarpsPerBlock = (DataWarpsPerGrid + gridDim.x-1) / gridDim.x; \
	const unsigned int DataCountPerBlock = (DataWarpsPerBlock << LOG2_WARP_SIZE); \
	const unsigned int WarpBorder = DataWarpsPerBlock % WarpsPerBlock; \
	const unsigned int WarpFactor = DataWarpsPerBlock / WarpsPerBlock; \
	const unsigned int warpIdx = (threadIdx.x >> LOG2_WARP_SIZE); \
	const unsigned int blockBeg = blockIdx.x * DataCountPerBlock; \
	const unsigned int blockEnd = min(blockBeg + DataCountPerBlock, count); \
	const unsigned int WarpSelect = (warpIdx < WarpBorder) ? 1 : 0; \
	const unsigned int WarpCount = WarpFactor + WarpSelect; \
	const unsigned int WarpOffset = warpIdx * WarpCount + WarpBorder * (1 - WarpSelect); \
	const unsigned int warpBeg = blockBeg + (WarpOffset << LOG2_WARP_SIZE); \
	const unsigned int warpEnd = min(warpBeg + (WarpCount << LOG2_WARP_SIZE), blockEnd); \
	__shared__ volatile unsigned int sdata[BlockSize * 2]; \
	__shared__ volatile unsigned int sdata1[BlockSize * 2];

SYNC_KERNEL_BEG(SCAN_WARPS_PER_BLOCK, scanKernel, unsigned int count,
	float benefitMin, float benefitMax, unsigned int* g_indices, const float* g_benefits, unsigned int* g_boundParams, unsigned int* g_tmpCounts, unsigned int* g_tmpCounts1
)
	SCAN_KERNEL_SETUP(count)

	scan1<WarpsPerBlock, BlockSize>(count,
		benefitMin, benefitMax, g_indices, g_benefits,
		g_boundParams, g_tmpCounts, g_tmpCounts1,
		sdata, sdata1,
		warpBeg, warpEnd);

	BLOCK_SYNC_BEGIN()

	scan2<WarpsPerBlock, BlockSize>(
		benefitMin, benefitMax, g_indices, g_benefits,
		g_boundParams, g_tmpCounts, g_tmpCounts1,
		sdata, sdata1,
		gridDim.x);
		
	BLOCK_SYNC_END()

	scan3<WarpsPerBlock, BlockSize>(count,
		benefitMin, benefitMax, g_indices, g_benefits,
		g_boundParams, g_tmpCounts, g_tmpCounts1,
		sdata, sdata1,
		warpBeg, warpEnd);

SYNC_KERNEL_END()

BOUND_KERNEL_BEG(SCAN_WARPS_PER_BLOCK, scan1Kernel,
	float benefitMin, float benefitMax, unsigned int* g_indices, const float* g_benefits, unsigned int* g_boundParams, unsigned int* g_tmpCounts, unsigned int* g_tmpCounts1
)
	SCAN_KERNEL_SETUP(_threadCount)

	scan1<WarpsPerBlock, BlockSize>(_threadCount,
		benefitMin, benefitMax, g_indices, g_benefits,
		g_boundParams, g_tmpCounts, g_tmpCounts1,
		sdata, sdata1,
		warpBeg, warpEnd);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SCAN_WARPS_PER_BLOCK, scan2Kernel,
	float benefitMin, float benefitMax, unsigned int* g_indices, const float* g_benefits, unsigned int* g_boundParams, unsigned int* g_tmpCounts, unsigned int* g_tmpCounts1,
	unsigned int gridSize
)
	__shared__ volatile unsigned int sdata[BlockSize * 2];
	__shared__ volatile unsigned int sdata1[BlockSize * 2];

	scan2<WarpsPerBlock, BlockSize>(
		benefitMin, benefitMax, g_indices, g_benefits,
		g_boundParams, g_tmpCounts, g_tmpCounts1,
		sdata, sdata1,
		gridSize);

BOUND_KERNEL_END()

BOUND_KERNEL_BEG(SCAN_WARPS_PER_BLOCK, scan3Kernel,
	float benefitMin, float benefitMax, unsigned int* g_indices, const float* g_benefits, unsigned int* g_boundParams, unsigned int* g_tmpCounts, unsigned int* g_tmpCounts1
)
	SCAN_KERNEL_SETUP(_threadCount)

	scan3<WarpsPerBlock, BlockSize>(_threadCount,
		benefitMin, benefitMax, g_indices, g_benefits,
		g_boundParams, g_tmpCounts, g_tmpCounts1,
		sdata, sdata1,
		warpBeg, warpEnd);

BOUND_KERNEL_END()
